#include "hip/hip_runtime.h"
#include "package/package.cuh"

/*
        0          1       2       3       4     5      6    7       8
./optimize_mdl data.bin mdl.bin out.bin echopes repeat sets score optimizer
	
	Score : meansquared, crossentropy
	Optimizer : sgd, moment, rmsprop, adam
*/

#define PERCENT_TEST_BATCHS 1.0

char * data_file = 0;
char * mdl_file = 0;
char * out_file = 0;
uint echopes = 1;
uint repeat = 1;
uint sets = 1;
uint score_algo = 0;
uint opti_algo = 0;
uint test_batchs = 1;
uint __select = 0;
uint test_all_batchs = 0;
uint echo = 0;

uint eq_pos(char * arg) {
	uint i=0;
	char c = arg[0];
	while (1) {
		if (c == '=')
			return i;
		if (c == '\0')
			ERR("No '=' in {%s}", arg);
		i++;
		c = arg[i];
	}
	return 0;
};

void analyse_args(int argc, char ** argv) {
	uint __ep_pos;

	uint _len;
	for (int i=1; i < argc; i++) {
		__ep_pos = eq_pos(argv[i]);
		argv[i][__ep_pos] = '\0';

		if (strcmp(argv[i], "data_file") == 0) {
			_len = strlen(argv[i] + __ep_pos+1);
			data_file = (char*)malloc(sizeof(char) * (_len + 1));
			strcpy(data_file, argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "mdl_file") == 0) {
			_len = strlen(argv[i] + __ep_pos+1);
			mdl_file = (char*)malloc(sizeof(char) * (_len + 1));
			strcpy(mdl_file, argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "out_file") == 0) {
			_len = strlen(argv[i] + __ep_pos+1);
			out_file = (char*)malloc(sizeof(char) * (_len + 1));
			strcpy(out_file, argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "echopes") == 0) {
			echopes = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "repeat") == 0) {
			repeat = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "sets") == 0) {
			sets = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "score_algo") == 0) {
			score_algo = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "opti_algo") == 0) {
			opti_algo = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "test_batchs") == 0) {
			test_batchs = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "select") == 0) {
			__select = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "test_all_batchs") == 0) {
			test_all_batchs = atoi(argv[i]+__ep_pos+1);
		} else if (strcmp(argv[i], "echo") == 0) {
			echo = atoi(argv[i]+__ep_pos+1);
		} else {
			ERR("What is %s ?", argv[i]);
		}
	}
};

uint find_min(float * arr, uint len) {
	uint __min_id = 0;

	for (uint i=1; i < len; i++)
		if (arr[__min_id] > arr[i])
			__min_id = i;

	return __min_id;
}

#define ECHOPES_PRINT 20

int main(int argc, char ** argv) {

	analyse_args(argc, argv);

	FILE * mdlfp = fopen(mdl_file, "rb");
	Mdl_t * mdl = mdl_fp_load(mdlfp);
	fclose(mdlfp);

	//// Load to Ram and Vram
	Data_t * data = data_open(data_file);
	data_cudmalloc(data);

	uint test_batchs = floor(PERCENT_TEST_BATCHS * data->batchs + 0.5);

	FILE * data_fp = fopen(data_file, "rb");

	//// Build Train_t and Opti_t
	if (sets == 0)
		ERR("sets can't be = to 0")
	Train_t * train = mk_train(mdl, data, sets);
	train_random_weights(train, rand()%10000);

	Opti_t * opti = opti_mk(train, score_algo, opti_algo);

	float set_score_tests[sets];

	uint start_seed;

	uint batch_train;
	
	//	On veut print l'échope que le program train, mais en Max ECHOPES_PRINT fois.
	//	Donc on le print tout les echopes/ECHOPES_PRINT fois. Sauf Si echopes < ECHOPES_PRINT.
	//	Dans ce cas on print echopes fois.
	uint _tmpt = ceil((float)echopes / ECHOPES_PRINT);	//comme round mais donne toujours au dessus

	for (uint lp=0; lp < echopes; lp++) {
		//Loop
		batch_train = rand() % data->batchs;

		//	Load a batch
		data_load_batch(data, data_fp, batch_train);
		data_cudamemcpy(data);

		train_set_input(train);

		//	Trainning Part
		for (uint i=0; i < repeat; i++) {
			//	Initialise correctly
			train_set_input(train);
			train_null_grad_meand(train);
	
			//	Forward and Backward
			start_seed = rand() % 100000;

			train_forward(train, start_seed);
			opti_dloss(opti);
			train_backward(train, start_seed);

			if (echo)
				train_print_meands(train);

			//	Optimize
			opti_opti(opti);
		}

		if (echopes < ECHOPES_PRINT || ((lp+1) % _tmpt) == 0) {
			printf("Echope : %i/%i [batch=%i]\n", lp, echopes, batch_train);

			/*for (uint s=0; s < sets; s++)
				set_score_tests[s] = 0;

			for (uint i=0; i < test_batchs; i++) {
				//	Select the best
				batch_train = rand() % data->batchs;
					
				data_load_batch(data, data_fp, batch_train);
				data_cudamemcpy(data);
				opti_loss(opti);
				
				for (uint s=0; s < sets; s++)
					set_score_tests[s] += opti->set_score[s] / test_batchs;
			}

			for (uint s=0; s < sets; s++) {
				printf("%i|\033[93m %f \033[0m\n", s, set_score_tests[s]);
			}*/

		}
	};

	printf("Echope : %i/%i [batch=%i]\n", echopes, echopes, batch_train);

	////	Compute Score
	for (uint s=0; s < sets; s++)
		set_score_tests[s] = 0;

	if (test_all_batchs) {
		for (uint i=0; i < data->batchs; i++) {
			data_load_batch(data, data_fp, i);
			data_cudamemcpy(data);
			opti_loss(opti);
			
			for (uint s=0; s < sets; s++)
				set_score_tests[s] += opti->set_score[s] / data->batchs;
		}
	} else {
		for (uint i=0; i < test_batchs; i++) {
			//	Select the best
			batch_train = rand() % data->batchs;
				
			data_load_batch(data, data_fp, batch_train);
			data_cudamemcpy(data);
			opti_loss(opti);
			
			for (uint s=0; s < sets; s++)
				set_score_tests[s] += opti->set_score[s] / test_batchs;
		}
	}

	printf("## Scores ##\n");
	for (uint s=0; s < sets; s++) {
		//set_score_tests[s] /= test_batchs;
		printf("|| %i | \033[93m %f \033[0m\n", s, set_score_tests[s]);
	}

	//	Take Best set
	uint best_set = find_min(set_score_tests, sets);//opti->podium[0];

	train_cpy_ws_to_mdl(train, best_set);

	mdlfp = fopen(out_file, "wb");
	mdl_fp_write(mdl, mdlfp);
	fclose(mdlfp);

	//	Free all to make a correct valgrind and juste to make all clean (and each malloc have to be freed)
	opti_free(opti);
	train_free(train);
	data_free(data);
	mdl_free(mdl);
};