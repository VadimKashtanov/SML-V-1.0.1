#include "package/optis/adam/head/adam.cuh"

float opti_adam_alpha = 1e-5;
float opti_adam_beta0 = 1e-5;
float opti_adam_beta1 = 1e-5;

//	Minimum echopes to test the potential
uint ADAM_min_echopes = 2;

void * ADAM_space_mk(Opti_t * opti) {
	float * m_d, * v_d;

	SAFE_CUDA(hipMalloc((void**)&m_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights));
	SAFE_CUDA(hipMalloc((void**)&v_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights));

	SAFE_CUDA(hipMemset(m_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights));
	SAFE_CUDA(hipMemset(v_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights));

	AdamData_t * ret = (AdamData_t*)malloc(sizeof(AdamData_t));

	ret->m_d = m_d;
	ret->v_d = v_d;

	ret->echope = 0;

	return (void*)ret;
};

void ADAM_free(Opti_t * opti) {
	SAFE_CUDA(hipFree(((AdamData_t*)opti->opti_space)->m_d))
	SAFE_CUDA(hipFree(((AdamData_t*)opti->opti_space)->v_d))
	free(((AdamData_t*)opti->opti_space));
};

void ADAM_set_one_arg(Opti_t * opti, char * name, char * value) {
	if (strcmp(name, "ALPHA") == 0) {
		opti_adam_alpha = atof(value);
	} else if (strcmp(name, "BETA0") == 0) {
		opti_adam_beta0 = atof(value);
	} else if (strcmp(name, "BETA1") == 0) {
		opti_adam_beta1 = atof(value);
	} else {
		ERR("What is %s (of value %s)", name, value);
	}
};

const char * ADAM_CONST_ARRAY[ADAM_CONSTS] = {"ALPHA", "BETA0", "BETA1"};
const uint ADAM_CONSTS_AMOUNT = ADAM_CONSTS;