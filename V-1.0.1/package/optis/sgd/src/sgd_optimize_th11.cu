#include "hip/hip_runtime.h"
#include "package/optis/sgd/head/sgd.cuh"

__global__
void sgd_kernel_th11(
	float sgd_alpha,
	uint weights, uint lines,
	float * weight, float * meand)
{
	uint w = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (w < weights)
		weight[set*weights + w] -= sgd_alpha * meand[set*weights + w] / lines;
};

void SGD_optimize(Opti_t * opti)
{
	SGDData_t * ret = (SGDData_t*)opti->opti_space;

	ret->echopes++;

	sgd_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_sgd_alpha,
		opti->train->mdl->weights, opti->train->data->lines,
		opti->train->_weight, opti->train->_meand
	);

	//opti_sgd_alpha *= ( 1 / ( 1 + 0.1/2000 * ret->echopes));

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//
//
//		Tester avec un probleme de beaucoup plus petite taille
//		Pas du Mnist, mais par exemple classifier 3 lettres A,B,C avec des images de taille 8x8
//		Pour kconvl 8x8->4x4->2x2->dot1d(4)->softmax(4)
//
//		1. Faire en python un petit programme qui ecrit les lettres en pixels avec l'IDLE
//		2. Faire un Data_t avec ces pixels
//		3. Cree le model simple et observer les trucs
//
//
//
//		Au lieux de fait un model qui predit plusieurs classes,
//		On train plusieur models d'une seule classe binaire.
//		Ex : On train le model qui predit un 0 ou un autre nombre que 0
//			puis On train le model qui predit un 1 ou un autre nombre que 1
//			puis le model qui predit un 2 ou un autre nombre que 2
//				..., jusqu'a 9
//		Apres on unie tout. On y ajoute un dot1->dot1d->softmax (en coupant le derniere dot1d->softmax des models precedants)
//		Bon ducoup ça train pas la diversité ensemble (des 0,1,2,3 .. n)
//		Mais ça va s'auto train juste apres tout seul.
//
//		(a la limite si les kernels bougent trop, on peut donner le resulats des model_precedant(input_qui_correspond) -> output)
//		(et le output sera mis dans le Data_t et puis plus tard, une fois train, on met ensemble tous les models)