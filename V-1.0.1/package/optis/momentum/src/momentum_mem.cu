#include "package/optis/momentum/head/momentum.cuh"

float opti_momentum_alpha = 1e-5;
float opti_momentum_moment = 1 - 1e-5;

//	Minimum echopes to test the potential
uint MOMENTUM_min_echopes = 2;

void * MOMENTUM_space_mk(Opti_t * opti) {
	float * ret_d;

	SAFE_CUDA(hipMalloc((void**)&ret_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	SAFE_CUDA(hipMemset(ret_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	return (void*)ret_d;
};

void MOMENTUM_free(Opti_t * opti) {
	SAFE_CUDA(hipFree((float*)opti->opti_space))
};

void MOMENTUM_set_one_arg(Opti_t * opti, char * name, char * value) {
	if (strcmp(name, "ALPHA") == 0) {
		opti_momentum_alpha = atof(value);
	} else if (strcmp(name, "MOMENT") == 0) {
		opti_momentum_moment = atof(value);
	} else {
		ERR("What is %s (of value %s)", name, value);
	}
};

const char * MOMENTUM_CONST_ARRAY[MOMENTUM_CONSTS] = {"ALPHA", "MOMENT"};
const uint MOMENTUM_CONSTS_AMOUNT = MOMENTUM_CONSTS;