#include "package/optis/rmsprop/head/rmsprop.cuh"

float opti_rmsprop_alpha = 1e-5;
float opti_rmsprop_beta = 1e-4;

//	Minimum echopes to test the potential
uint RMSPROP_min_echopes = 2;

void * RMSPROP_space_mk(Opti_t * opti) {
	float * v0_d;//, * v1_d;

	SAFE_CUDA(hipMalloc((void**)&v0_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	//SAFE_CUDA(hipMalloc((void**)&v1_d, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	SAFE_CUDA(hipMemset(v0_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))
	//SAFE_CUDA(hipMemset(v1_d, 0, sizeof(float) * opti->train->sets * opti->train->mdl->weights))

	//RMSprop_data_t * ret = (RMSprop_data_t*)malloc(sizeof(RMSprop_data_t));

	//ret->v0_d = v0_d;
	//ret->v1_d = v1_d;

	//return (void*)ret;
	return (void*)v0_d;
};

void RMSPROP_free(Opti_t * opti) {
	//SAFE_CUDA(hipFree((RMSprop_data_t*)opti->opti_space->v0_d))
	//SAFE_CUDA(hipFree((RMSprop_data_t*)opti->opti_space->v1_d))
	//free((RMSprop_data_t*)opti->opti_space);
	SAFE_CUDA(hipFree((float*)opti->opti_space))
};

void RMSPROP_set_one_arg(Opti_t * opti, char * name, char * value) {
	if (strcmp(name, "ALPHA") == 0) {
		opti_rmsprop_alpha = atof(value);
	} else if (strcmp(name, "BETA") == 0) {
		opti_rmsprop_beta = atof(value);
	} else {
		ERR("What is %s (of value %s)", name, value);
	}
};

const char * RMSPROP_CONST_ARRAY[RMSPROP_CONSTS] = {"ALPHA", "BETA"};
const uint RMSPROP_CONSTS_AMOUNT = RMSPROP_CONSTS;