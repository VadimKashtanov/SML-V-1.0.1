#include "hip/hip_runtime.h"
#include "package/optis/rmsprop/head/rmsprop.cuh"

__global__
void RMSPROP_kernel_th11(
	float alpha, float beta,
	uint weights, uint lines,
	float * v, float * weight, float * meand)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	float _v, dw;

	if (x < weights) {
		dw = meand[set*weights + x] / lines;
		_v = beta * v[set*weights + x] + (1-beta) * pow(dw,2);
		v[set*weights + x] = _v;
		weight[set*weights + x] -= alpha * dw * pow(_v + 1e-8, -0.5);	//eta = 1e-8
	}
};

void RMSPROP_optimize(Opti_t * opti)
{
	RMSPROP_kernel_th11<<<dim3(KERN_DIV(opti->train->mdl->weights, 16), opti->train->sets),dim3(16,1)>>>(
		opti_rmsprop_alpha, opti_rmsprop_beta,
		opti->train->mdl->weights, opti->train->data->lines,
		(float*)opti->opti_space, opti->train->_weight, opti->train->_meand
	);
	SAFE_CUDA(hipPeekAtLastError());
};