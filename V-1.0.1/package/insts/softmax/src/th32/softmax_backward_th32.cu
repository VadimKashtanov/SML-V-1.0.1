#include "hip/hip_runtime.h"
#include "package/insts/softmax/head/softmax.cuh"

/*
for i in range(_len):
	err = errors[i]
			
	for j in range(_len):
		yi = var[i]
		yj = var[j]

		if i == j:
			grad[j] += err * yi * (1 - yi)
		else:
			grad[j] += - err * yi * yj

Sauf que j'ai inverser i et j, pour ne pas faire de atomicAdd(&) beaucoup trop

for j in range(_len):
	_grad = 0
	for i in range(_len):
		yi = var[i]
		yj = var[j]
			
		if i == j:
			_grad += errors[i] * yi * (1 - yi)
		else:
			_grad += - errors[i] * yi * yj
	grad[j] = _grad
*/

__global__
void softmax_backward_th32(
	uint len,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart,
	uint sets,
	float * var, float * grad)
{
	uint j = threadIdx.x;
	uint set = blockIdx.x;

	if (j < len) {
		float _grad = 0;

		__shared__ float _y[32];
		__shared__ float _err[32];

		_y[j] = var[time*sets*total + set*total + ystart + j];
		_err[j] = grad[time*sets*total + set*total + ystart + j];

		__syncthreads();

		//	Au lieu de faire une boucle avec un if a chaque fois, on fait 2 boucle et le seul cas quand i==j
		//Mais de toute façon ça reste
		//	(i==j) err[j] * yj * (1 - yj)
		//	(i!=i) err[i] * yi * yj

		for (uint i=0; i < j; i++)
			_grad += -_err[i] * _y[i] * _y[j];

		//err[j] et pas err[i] car i == j (sauf que `i` est utilisé que dans la boucle for)
		_grad += _err[j] * _y[j] * (1-_y[j]);	//car i==j

		for (uint i=j+1; i < len; i++)
			_grad += -_err[i] * _y[i] * _y[j];

		//on met dans le grad
		grad[time*sets*total + set*total + istart + j] = _grad;	//input_start car la on fait le gradient de l'input (car l'erreur (grad output) est déjà calculé)
	}
}