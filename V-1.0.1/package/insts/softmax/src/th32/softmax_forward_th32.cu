#include "hip/hip_runtime.h"
#include "package/insts/softmax/head/softmax.cuh"

__global__
void softmax_forward_th32(
	uint len,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart,
	uint sets,
	float * var)
{
	uint pos = threadIdx.x;
	uint set = blockIdx.x;

	if (pos < len) {
		float exped = exp(var[time*sets*total + set*total + istart + pos]);
		__shared__ float sum;
		if (pos == 0) sum = 0;
		__syncthreads();
		atomicAdd(&sum, exped);
		__syncthreads();
		var[time*sets*total + set*total + ystart + pos] = exped / sum;
	}
};