#include "hip/hip_runtime.h"
#include "package/insts/dot2drecurent/head/dot2drecurent.cuh"

//			   0  1    2   3    4      5       6    7			8       9
//Arguments = [Ax,Ay, At, Bx, active, istart,ystart,wstart,locdstart, drate]

void dot2drecurent_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[3] == 0) raise(SIGINT);
	if (param[4] >= 4) raise(SIGINT);
	if (param[9] >100) raise(SIGINT);
};

void dot2drecurent_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],	\
		 Ay=mdl->param[inst][1],	\
		 At=mdl->param[inst][2],	\
		 Bx=mdl->param[inst][3],	\
		 activ=mdl->param[inst][4],\
		 istart=mdl->param[inst][5],\
		 ystart=mdl->param[inst][6],\
		 wstart=mdl->param[inst][7];

	uint total = mdl->total;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float _tmp;

	//uint Apos, Wpos;

	for (uint y=0; y < Ay; y++) {
		for (uint x=0; x < Bx; x++) {
			_tmp = 0;

			if (time >= At) {	//oui il y a des constantes, mais je les bougerais un jours

				for (uint i=0; i < Ax; i++) {
					_tmp += var[(time-At)*total + istart + y*Ax + i] * weight[wstart + i*Bx + y];
				}
			}

			_tmp += weight[wstart + Ax*Bx + (y*Bx + x)];

			if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
			else if (activ == 1) _tmp = tanh(_tmp);
			else if (activ == 2) _tmp = exp(-_tmp*_tmp);
			else _tmp *= (_tmp > 0);
	
			var[time*total + ystart + (y*Bx + x)] = _tmp;
		}
	}
};

void dot2drecurent_use(Use_t * use, uint inst, uint time) {
	dot2drecurent_use_call_mode_th11(use, inst, time);
};

void dot2drecurent_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2drecurent_forward_call_mode_th11(train, inst, time, start_seed);
};

void dot2drecurent_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	dot2drecurent_backward_call_mode_th11(train, inst, time, start_seed);
};