#include "hip/hip_runtime.h"
#include "package/insts/dot2drecurent/head/dot2drecurent.cuh"

__global__
void dot2drecurent_use_th11(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	float _tmp;

	if (y < Ay && x < Bx) {

		_tmp = 0;

		for (uint i=0; i < Ax; i++) {
			_tmp += var[(time-At)*total + istart + y*Ax + i] * weight[wstart + i*Bx + y];
		}

		_tmp += weight[wstart + Bx*Ax + (y*Bx + x)];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
		else if (activ == 1) _tmp = tanh(_tmp);
		else if (activ == 2) _tmp = exp(-_tmp*_tmp);
		else _tmp *= (_tmp > 0);

		var[time*total + ystart + (y*Bx + x)] = _tmp;
	}
};

__global__
void dot2drecurent_use_th11_NegativLine(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	float _tmp;

	if (y < Ay && x < Bx) {

		_tmp = weight[wstart + Bx*Ax + (y*Bx + x)];	//==wstart + Ax*Yx + y      car on a deja +y, et on a += Ax*Yx (for i<Ax) {+=Yx}

		if (activ == 0)	_tmp = 1 / (1 + exp(-_tmp));
		else if (activ == 1) _tmp = tanh(_tmp);
		else if (activ == 2) _tmp = exp(-_tmp*_tmp);
		else _tmp *= (_tmp > 0);

		var[time*total + ystart + (y*Bx + x)] = _tmp;
	}
};