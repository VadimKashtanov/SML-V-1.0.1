#include "hip/hip_runtime.h"
#include "package/insts/dot2drecurent/head/dot2drecurent.cuh"

__global__
void dot2drecurent_backward_th11(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float dlds;
	uint Apos, Wpos;

	if (y < Ay && x < Bx) {
		dlds = grad[time*sets*total + set*total + ystart + (y*Bx + x)] * locd[time*locdsize*sets + set*locdsize + locdstart + (y*Bx + x)];

		for (uint i=0; i < Ax; i++) {
			Apos = (time-At)*total*sets + set*total + istart + y*Ax + i;
			Wpos = wsize*set + wstart + y + i*Bx;

			if (pseudo_randomf(Apos*seed) >= drop_rate) {
				atomicAdd(grad + Apos, weight[Wpos] * dlds);
				atomicAdd(meand + Wpos, var[Apos] * dlds);
			}
		}

		meand[wsize*set + wstart + Bx*Ax + (y*Bx + x)] += dlds;
	}
}


__global__
void dot2drecurent_backward_th11_NegativLine(
	uint Ax, uint Ay, uint At, uint Bx,
	uint activ,
	uint time,
	uint istart, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float dlds;

	if (y < Ay && x < Bx) {
		dlds = grad[time*sets*total + set*total + ystart + (y*Bx + x)] * locd[time*locdsize*sets + set*locdsize + locdstart + (y*Bx + x)];

		meand[wsize*set + wstart + Bx*Ax + (y*Bx + x)] += dlds;
	}
}