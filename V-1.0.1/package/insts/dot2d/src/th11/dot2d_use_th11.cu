#include "hip/hip_runtime.h"
#include "package/insts/dot2d/head/dot2d.cuh"

__global__
void dot2d_use_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint activ,
	uint time,
	uint vars,
	uint input_start, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint Yx = threadIdx.x + blockIdx.x*blockDim.x,	\
		 Yy = threadIdx.y + blockIdx.y*blockDim.y;	\

	uint Apos = time*vars + input_start + Yy*Ax;
	uint Bpos = wstart + Yx;

	float sum = 0;
	for (uint i=0; i < Ax; i++) {
		sum += var[Apos] * weight[Bpos];
		Apos++;
		Bpos += Bx; 
	}
	sum += weight[wstart + Bx*Ax + Yy*Bx + Yx];

	if (activ == 0) sum = 1 / (1 + exp(-sum));
	else if (activ == 1) sum = tanh(sum);
	else if (activ == 2) sum = exp(-pow(sum,2));
	else if (activ == 3) sum = sum*(sum >= 0);
	//else _tmp = tmp

	var[time*vars + ystart + Yy*Bx + Yx] = sum;
};