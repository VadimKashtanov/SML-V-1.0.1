#include "package/insts/sum/head/sum.cuh"

void sum_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;

	uint size   = mdl->param[inst][0],	\
		 items  = mdl->param[inst][1],	\
		 istart = mdl->param[inst][2],	\
		 ystart = mdl->param[inst][3];

	sum_use_th11<<<dim3(KERN_DIV(size,32)),dim3(32)>>>(
		size, items,
		time,
		mdl->total,
		istart, ystart,
		use->var_d);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//======================== Train_t =======================

//-------------------------- forward ---------------------

void sum_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint size   = mdl->param[inst][0],	\
		 items  = mdl->param[inst][1],	\
		 istart = mdl->param[inst][2],	\
		 ystart = mdl->param[inst][3];

	sum_forward_th11<<<dim3(KERN_DIV(size,32), train->sets),dim3(32, 1)>>>(
		size, items,
		time,
		mdl->total, mdl->locds,
		istart, ystart,
		train->sets,
		train->_var);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//-------------------------- backward ---------------------

void sum_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint size   = mdl->param[inst][0],	\
		 items  = mdl->param[inst][1],	\
		 istart = mdl->param[inst][2],	\
		 ystart = mdl->param[inst][3];

	sum_backward_th11<<<dim3(KERN_DIV(size,32), train->sets),dim3(32, 1)>>>(
		size, items,
		time,
		mdl->total, mdl->locds,
		istart, ystart,
		train->sets,
		train->_var, train->_grad);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};