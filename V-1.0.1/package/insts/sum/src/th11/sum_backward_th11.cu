#include "hip/hip_runtime.h"
#include "package/insts/sum/head/sum.cuh"

__global__
void sum_backward_th11(
	uint size, uint items,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart,
	uint sets,
	float * var, float * grad)
{
	uint pos = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (pos < size) {
		float dlds = grad[time*sets*total + set*total + ystart + pos];

		for (uint j = 0; j < items; j++) {
			grad[time*sets*total + set*total + istart + j*size + pos] += dlds;
		}
	}
}