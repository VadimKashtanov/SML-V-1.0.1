#include "hip/hip_runtime.h"
#include "package/insts/sum/head/sum.cuh"

__global__
void sum_forward_th11(
	uint size, uint items,
	uint time,
	uint total, uint lsize,
	uint istart, uint ystart,
	uint sets,
	float * var)
{
	uint pos = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (pos < size) {
		float _sum = 0;

		for (uint j = 0; j < items; j++) {
			_sum += var[time*sets*total + set*total + istart + j*size + pos];
		}

		var[time*sets*total + set*total + ystart + pos] = _sum;
	}
};