#include "hip/hip_runtime.h"
#include "package/insts/sum/head/sum.cuh"

__global__
void sum_use_th11(
	uint size, uint items,
	uint time,
	uint total,
	uint istart, uint ystart,
	float * var)
{
	uint pos = threadIdx.x + blockIdx.x * blockDim.x;

	if (pos < size) {
		float _sum = 0;

		for (uint j = 0; j < items; j++) {
			_sum += var[time*total + istart + j*size + pos];
		}

		var[time*total + ystart + pos] = _sum;
	}
}