#include "package/insts/kconvl2d/head/kconvl2d.cuh"

void kconvl2d_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	//
	Mdl_t * mdl = use->mdl;
	uint total = mdl->total;
	uint weights = mdl->weights;
	//uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Kx = param[2];
	uint Ky = param[3];
	uint n0 = param[4];
	uint n1 = param[5];
	uint strideX = param[6];
	uint strideY = param[7];
	uint paddingX = param[8];
	uint paddingY = param[9];
	uint activ = param[10];
	uint istart = param[11];
	uint ystart = param[12];
	uint wstart = param[13];
	//uint locdstart = param[14];
	//uint drop_rate = param[15];

	uint Yx = (Ax - 2*paddingX) / strideX;
	uint Yy = (Ay - 2*paddingY) / strideY;

	kconvl2d_use_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), n1), dim3(32,32, 1)>>>(
		Yx, Yy,
		n0, n1, Ax, Ay,
		Kx, Ky,
		strideX, strideY,
		paddingX, paddingY,
		activ,						
		time,
		total, weights,
		istart, wstart, ystart,
		use->var_d, use->weight_d);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}

void kconvl2d_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Kx = param[2];
	uint Ky = param[3];
	uint n0 = param[4];
	uint n1 = param[5];
	uint strideX = param[6];
	uint strideY = param[7];
	uint paddingX = param[8];
	uint paddingY = param[9];
	uint activ = param[10];
	uint istart = param[11];
	uint ystart = param[12];
	uint wstart = param[13];
	uint locdstart = param[14];
	uint drop_rate = param[15];

	uint Yx = (Ax - 2*paddingX) / strideX;
	uint Yy = (Ay - 2*paddingY) / strideY;

	for (uint _set=0; _set < train->sets; _set++) {
		kconvl2d_forward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), n1), dim3(32,32, 1)>>>(
			Yx, Yy,
			n0, n1, Ax, Ay,
			Kx, Ky,
			strideX, strideY,
			paddingX, paddingY,
			activ,
			time,
			total, weights, locds,
			istart, wstart, ystart, locdstart,
			start_seed, (float)drop_rate / 100,
			_set, train->sets,
			train->_var, train->_weight, train->_locd);
		hipDeviceSynchronize();
		SAFE_CUDA(hipPeekAtLastError());
	}
}

void kconvl2d_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Kx = param[2];
	uint Ky = param[3];
	uint n0 = param[4];
	uint n1 = param[5];
	uint strideX = param[6];
	uint strideY = param[7];
	uint paddingX = param[8];
	uint paddingY = param[9];
	uint activ = param[10];
	uint istart = param[11];
	uint ystart = param[12];
	uint wstart = param[13];
	uint locdstart = param[14];
	uint drop_rate = param[15];

	uint Yx = (Ax - 2*paddingX) / strideX;
	uint Yy = (Ay - 2*paddingY) / strideY;

	for (uint _set=0; _set < train->sets; _set++) {
		kconvl2d_backward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), n1), dim3(32,32, 1)>>>(
			Yx, Yy,
			n0, n1, Ax, Ay,
			Kx, Ky,
			strideX, strideY,
			paddingX, paddingY,
			activ,						
			time,
			total, weights, locds,
			istart, wstart, ystart, locdstart,
			start_seed, (float)drop_rate / 100,
			_set, train->sets,
			train->_var, train->_weight, train->_locd,
			train->_grad, train->_meand);
		hipDeviceSynchronize();
		SAFE_CUDA(hipPeekAtLastError());
	}
}