#include "hip/hip_runtime.h"
#include "package/insts/kconvl2d/head/kconvl2d.cuh"

void kconvl2d_check(uint * param) {
	uint Ax = param[0];
	uint Ay = param[1];
	uint Kx = param[2];
	uint Ky = param[3];
	uint n0 = param[4];
	uint n1 = param[5];
	uint strideX = param[6];
	uint strideY = param[7];
	//uint paddingX = param[8];
	//uint paddingY = param[9];
	uint activ = param[10];
	//uint input_start = param[11];
	//uint ystart = param[12];
	//uint wstart = param[13];
	//uint locdstart = param[14];
	uint drop_rate = param[15];

	assert(Kx % 2 != 0);
	assert(Ky % 2 != 0);
	assert(activ < 5);
	assert(strideX > 0);
	assert(strideY > 0);
	assert(Ax > 0);
	assert(Ay > 0);
	assert(n0 > 0);
	assert(n1 > 0);
	assert(Ax % strideX == 0);
	assert(Ay % strideY == 0);
	assert(drop_rate <= 100);
};

void kconvl2d_cpu(Cpu_t * cpu, uint inst, uint time) {
	//
	Mdl_t * mdl = cpu->mdl;
	uint total = mdl->total;

	//
	float * var = cpu->var;
	float * weight = mdl->weight;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Kx = param[2];
	uint Ky = param[3];
	uint n0 = param[4];
	uint n1 = param[5];
	uint strideX = param[6];
	uint strideY = param[7];
	uint paddingX = param[8];
	uint paddingY = param[9];
	uint activ = param[10];
	uint istart = param[11];
	uint ystart = param[12];
	uint wstart = param[13];
	//uint locdstart = param[14];
	//uint drop_rate = param[15];

	//
	uint ker_radiusX = (Kx-1)/2;
	uint ker_radiusY = (Ky-1)/2;

	uint Yx = (Ax - 2*paddingX) / strideX;
	uint Yy = (Ay - 2*paddingY) / strideY;

	float _sum;
	uint _pixelpos, _kernelpos;

	uint start_ker_x, end_ker_x, start_ker_y, end_ker_y;

	for (uint _n1=0; _n1 < n1; _n1++) {

		for (uint y=paddingY; y < Ay - paddingY; y += strideY) {
			for (uint x=paddingX; x < Ax - paddingX; x += strideX) {

				_sum = 0;

				start_ker_x = (x >= ker_radiusX) ? 0 : (ker_radiusX - x);	//max((ker_radius-x), 0)  it's kind of distance beetwin kernel border and image border
				start_ker_y = (y >= ker_radiusY) ? 0 : (ker_radiusY - y);

				end_ker_x = (x < (Ax - ker_radiusX)) ? Kx : (Kx - ((x+ker_radiusX) - Ax+1));
				end_ker_y = (y < (Ay - ker_radiusY)) ? Ky : (Ky - ((y+ker_radiusY) - Ay+1));

				for (uint _n0=0; _n0 < n0; _n0++) {
					for (uint ker_y=start_ker_y; ker_y < end_ker_y; ker_y++) {
						for (uint ker_x=start_ker_x; ker_x < end_ker_x; ker_x++) {
							_pixelpos = time*total + istart + _n0*Ax*Ay + (y+ker_y-ker_radiusY)*Ax + (x+ker_x-ker_radiusX);
							_kernelpos = wstart + _n1*Kx*Ky*n0 + _n0*Kx*Ky + (ker_y*Kx + ker_x);

							_sum += var[_pixelpos] * weight[_kernelpos];
						}
					}
				}

				_pixelpos = _n1*Yx*Yy + ((y-paddingY)/strideY)*Yx + ((x-paddingX)/strideX);
				_sum += weight[wstart + n1*n0*Kx*Ky + _pixelpos];

				if (activ == 0) _sum = 1 / (1 + exp(-_sum));
				else if (activ == 1) _sum = tanh(_sum);
				else if (activ == 2) _sum = exp(-_sum * _sum);
				else if (activ == 3) _sum = _sum * (_sum > 0);
				//else _tmp = tmp
				
				var[time*total + ystart + _pixelpos] = _sum;
			}
		}
	}
};

void kconvl2d_use(Use_t * use, uint inst, uint time) {
	//the only mode is th11
	kconvl2d_use_call_mode_th11(use, inst, time);
};

void kconvl2d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	kconvl2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void kconvl2d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	kconvl2d_backward_call_mode_th11(train, inst, time, start_seed);
};