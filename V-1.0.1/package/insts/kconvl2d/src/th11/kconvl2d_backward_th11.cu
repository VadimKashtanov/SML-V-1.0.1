#include "hip/hip_runtime.h"
#include "package/insts/kconvl2d/head/kconvl2d.cuh"

/*
	Each kernel backwards one pixel for each channels (n1 pixels)

	<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), _n1), dim3(32,32, 1)>>>
*/

__global__
void kconvl2d_backward_th1x1(
	uint Yx, uint Yy,
	uint n0, uint n1, uint Ax, uint Ay,
	uint Kx, uint Ky,
	uint strideX, uint strideY,
	uint paddingX, uint paddingY,
	uint activ,
	uint time,
	uint total, uint weights, uint locds,
	uint istart, uint wstart, uint ystart, uint lstart,
	uint seed, float drop_rate,
	uint _set, uint sets,
	float * var, float * weight, float * locd,
	float * grad, float * meand)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _n1 = blockIdx.z;

	uint x = _Yx*strideX + paddingX;
	uint y = _Yy*strideY + paddingY;

	if (_Yx < Yx && _Yy < Yy) {

		uint _pixelpos, _kernelpos;

		uint ker_radiusX = (Kx-1)/2;
		uint ker_radiusY = (Ky-1)/2;

		_pixelpos = _n1*Yx*Yy + ((y-paddingY)/strideY)*Yx + ((x-paddingX)/strideX);

		float dlds = grad[time*sets*total + _set*total + ystart + _pixelpos] * locd[time*sets*locds + _set*locds + lstart + _pixelpos];

		meand[_set*weights + wstart + n1*n0*Kx*Ky + _pixelpos] += dlds;

		uint start_ker_x = (x >= ker_radiusX) ? 0 : (ker_radiusX - x);	//max((ker_radius-x), 0)  it's kind of distance beetwin kernel border and image border
		uint start_ker_y = (y >= ker_radiusY) ? 0 : (ker_radiusY - y);

		uint end_ker_x = (x < (Ax - ker_radiusX)) ? Kx : (Kx - ((x+ker_radiusX) - Ax+1));
		uint end_ker_y = (y < (Ay - ker_radiusY)) ? Ky : (Ky - ((y+ker_radiusY) - Ay+1));

		for (uint _n0=0; _n0 < n0; _n0++) {
			for (uint ker_y=start_ker_y; ker_y < end_ker_y; ker_y++) {
				for (uint ker_x=start_ker_x; ker_x < end_ker_x; ker_x++) {
					_pixelpos = time*sets*total + _set*total + istart + _n0*Ax*Ay + (y+ker_y-ker_radiusY)*Ax + (x+ker_x-ker_radiusX);
					_kernelpos = _set*weights + wstart + _n1*Kx*Ky*n0 + _n0*Kx*Ky + (ker_y*Kx + ker_x);

					atomicAdd(&grad[_pixelpos], dlds * weight[_kernelpos]);
					atomicAdd(&meand[_kernelpos], dlds * var[_pixelpos]);

				}
			}
		}
	}
}