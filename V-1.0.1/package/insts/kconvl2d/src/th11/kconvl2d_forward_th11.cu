#include "hip/hip_runtime.h"
#include "package/insts/kconvl2d/head/kconvl2d.cuh"

/*
	Each kernel compute one pixel for each channels (n1 pixels)

	<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), _n1), dim3(32,32, 1)>>>
*/

__global__
void kconvl2d_forward_th1x1(
	uint Yx, uint Yy,
	uint n0, uint n1, uint Ax, uint Ay,
	uint Kx, uint Ky,
	uint strideX, uint strideY,
	uint paddingX, uint paddingY,
	uint activ,						
	uint time,
	uint total, uint weights, uint locds,
	uint istart, uint wstart, uint ystart, uint lstart,
	uint seed, float drop_rate,
	uint _set, uint sets,
	float * var, float * weight, float * locd)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _n1 = blockIdx.z;

	uint x = _Yx*strideX + paddingX;
	uint y = _Yy*strideY + paddingY;

	if (_Yx < Yx && _Yy < Yy) {
		uint ker_radiusX = (Kx-1)/2;
		uint ker_radiusY = (Ky-1)/2;

		uint _pixelpos, _kernelpos;

		float _sum = 0;

		uint start_ker_x = (x >= ker_radiusX) ? 0 : (ker_radiusX - x);	//max((ker_radius-x), 0)  it's kind of distance beetwin kernel border and image border
		uint start_ker_y = (y >= ker_radiusY) ? 0 : (ker_radiusY - y);

		uint end_ker_x = (x < (Ax - ker_radiusX)) ? Kx : (Kx - ((x+ker_radiusX) - Ax+1));
		uint end_ker_y = (y < (Ay - ker_radiusY)) ? Ky : (Ky - ((y+ker_radiusY) - Ay+1));

		for (uint _n0=0; _n0 < n0; _n0++) {
			for (uint ker_y=start_ker_y; ker_y < end_ker_y; ker_y++) {
				for (uint ker_x=start_ker_x; ker_x < end_ker_x; ker_x++) {
					_pixelpos = time*sets*total + _set*total + istart + _n0*Ax*Ay + (y+ker_y-ker_radiusY)*Ax + (x+ker_x-ker_radiusX);
					_kernelpos = _set*weights + wstart + _n1*Kx*Ky*n0 + _n0*Kx*Ky + (ker_y*Kx + ker_x);

					_sum += var[_pixelpos] * weight[_kernelpos];
				}
			}
		}

		_pixelpos = _n1*Yx*Yy + ((y-paddingY)/strideY)*Yx + ((x-paddingX)/strideX);
		_sum += weight[_set*weights + wstart + n1*n0*Kx*Ky + _pixelpos];

		float __locd;

		if (activ == 0) {
			_sum = 1 / (1 + exp(-_sum));
			__locd = _sum*(1 - _sum);	//f'(x) = f(x)(1 - f(x))
		} else if (activ == 1) {
			_sum = tanh(_sum);
			__locd = 1 - _sum*_sum;		//f'(x) = 1 - tanh(x)^2
		} else if (activ == 2) {
			__locd = _sum;
			_sum = exp(-pow(_sum,2));
			__locd = -2*__locd*_sum;	//f'(x) = -2x*e^(-x^2)
		} else if (activ == 3) {
			__locd = (_sum >= 0);
			_sum = _sum*__locd;
		} else if (activ == 4) {
			__locd = 1;
		}

		var[time*sets*total + _set*total + ystart + _pixelpos] = _sum;		//same assembler than putting it in if/else structure
		locd[time*sets*locds + _set*locds + lstart + _pixelpos] = __locd;
	}
};