#include "hip/hip_runtime.h"
#include "package/insts/kconvl2d/head/kconvl2d.cuh"

/*
	Each kernel compute one pixel for each channels (n1 pixels)

	<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), _n1), dim3(32,32, 1)>>>
*/

__global__
void kconvl2d_use_th1x1(
	uint Yx, uint Yy,
	uint n0, uint n1, uint Ax, uint Ay,
	uint Kx, uint Ky,
	uint strideX, uint strideY,
	uint paddingX, uint paddingY,
	uint activ,
	uint time,
	uint total, uint wsize,
	uint istart, uint wstart, uint ystart,
	float * var, float * weight)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;	//pour chaque pixel dans Input
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;	//donc on va projeter ce pixel sur Input avec (x,y) 
	uint _n1 = blockIdx.z;

	//	Projection du Centre de la convolution (_Yx,Yy) sur input
	uint x = _Yx*strideX + paddingX;
	uint y = _Yy*strideY + paddingY;

	if (_Yx < Yx && _Yy < Yy) {

		uint _pixelpos, _kernelpos;

		uint ker_radiusX = (Kx-1)/2;
		uint ker_radiusY = (Ky-1)/2;

		float _sum = 0;

		uint start_ker_x = (x >= ker_radiusX) ? 0 : (ker_radiusX - x);	//max((ker_radius-x), 0)  it's kind of distance beetwin kernel border and image border
		uint start_ker_y = (y >= ker_radiusY) ? 0 : (ker_radiusY - y);

		uint end_ker_x = (x < (Ax - ker_radiusX)) ? Kx : (Kx - ((x+ker_radiusX) - Ax+1));
		uint end_ker_y = (y < (Ay - ker_radiusY)) ? Ky : (Ky - ((y+ker_radiusY) - Ay+1));

		for (uint _n0=0; _n0 < n0; _n0++) {
			for (uint ker_y=start_ker_y; ker_y < end_ker_y; ker_y++) {
				for (uint ker_x=start_ker_x; ker_x < end_ker_x; ker_x++) {
					_pixelpos = time*total + istart + _n0*Ax*Ay + (y+ker_y-ker_radiusY)*Ax + (x+ker_x-ker_radiusX);
					_kernelpos = wstart + _n1*Kx*Ky*n0 + _n0*Kx*Ky + (ker_y*Kx + ker_x);

					_sum += var[_pixelpos] * weight[_kernelpos];
				}
			}
		}

		_pixelpos = _n1*Yx*Yy + ((y-paddingY)/strideY)*Yx + ((x-paddingX)/strideX);
		_sum += weight[wstart + n1*n0*Kx*Ky + _pixelpos];

		if (activ == 0) _sum = 1 / (1 + exp(-_sum));
		else if (activ == 1) _sum = tanh(_sum);
		else if (activ == 2) _sum = exp(-_sum * _sum);
		else if (activ == 3) _sum = _sum * (_sum > 0);
		//else _tmp = tmp

		var[time*total + ystart + _pixelpos] = _sum;
	}
}