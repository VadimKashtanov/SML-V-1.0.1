#include "hip/hip_runtime.h"
#include "package/insts/pool2daverage/head/pool2daverage.cuh"

__global__
void pool2daverage_forward_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,						
	uint time,
	uint total, uint locds,
	uint istart, uint ystart,
	uint sets,
	float * var, float * locd)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _set = blockIdx.z;

	if (_Yx < Yx && _Yy < Yy) {
		float _sum = 0;

		for (uint _y=0; _y < Ypool; _y++) {
			for (uint _x=0; _x < Xpool; _x++) {
				_sum += var[time*sets*total + _set*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)];
			}
		}

		var[time*sets*total + _set*total + ystart + _Yy*Yx + _Yx] = _sum / (Xpool * Ypool);
	};
}