#include "hip/hip_runtime.h"
#include "package/insts/pool2daverage/head/pool2daverage.cuh"

__global__
void pool2daverage_use_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,
	uint time,
	uint total,
	uint istart, uint ystart,
	float * var)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;

	if (_Yx < Yx && _Yy < Yy) {
		float _sum = 0;

		for (uint _y=0; _y < Ypool; _y++) {
			for (uint _x=0; _x < Xpool; _x++) {
				_sum += var[time*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)];
			}
		}

		var[time*total + ystart + _Yy*Yx + _Yx] = _sum / (Xpool * Ypool);
	};
};