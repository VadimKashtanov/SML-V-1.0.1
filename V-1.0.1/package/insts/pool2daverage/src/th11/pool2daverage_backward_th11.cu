#include "hip/hip_runtime.h"
#include "package/insts/pool2daverage/head/pool2daverage.cuh"

__global__
void pool2daverage_backward_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,
	uint time,
	uint total, uint locds,
	uint istart, uint ystart,
	uint sets,
	float * var, float * locd,
	float * grad)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _set = blockIdx.z;

	if (_Yx < Yx && _Yy < Yy) {
		float dl_dpoolmax = grad[time*sets*total + _set*total + ystart + _Yy*Yx + _Yx] / (Xpool * Ypool);

		for (uint _y=0; _y < Ypool; _y++) {
			for (uint _x=0; _x < Xpool; _x++) {
				atomicAdd(&grad[time*sets*total + _set*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)], dl_dpoolmax);
			}
		}

		//	Si j'ajoute Stride, il faudra conserver le atomicAdd
		//	Sans le stride je peux en réalité mettre que += car chaque pixel de l'input n'est utilisé que dans un seul pool
	}
}