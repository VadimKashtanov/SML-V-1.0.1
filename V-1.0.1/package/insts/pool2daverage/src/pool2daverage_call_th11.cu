#include "package/insts/pool2daverage/head/pool2daverage.cuh"

void pool2daverage_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	//
	Mdl_t * mdl = use->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	//uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];
	//uint locdstart = param[6];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2daverage_use_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32)), dim3(32,32)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,
		time,
		total,
		istart, ystart,
		use->var_d);

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}

void pool2daverage_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2daverage_forward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), train->sets), dim3(32,32, 1)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,				
		time,
		total, locds,
		istart, ystart,
		train->sets,
		train->_var, train->_locd);
	
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}

void pool2daverage_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2daverage_backward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), train->sets), dim3(32,32, 1)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,
		time,
		total, locds,
		istart, ystart,
		train->sets,
		train->_var, train->_locd,
		train->_grad);

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}