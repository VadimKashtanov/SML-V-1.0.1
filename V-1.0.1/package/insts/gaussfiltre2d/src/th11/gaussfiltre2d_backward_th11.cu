#include "hip/hip_runtime.h"
#include "package/insts/gaussfiltre2d/head/gaussfiltre2d.cuh"

__global__
void gaussfiltre2d_backward_th11(
	uint X, uint Y,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	float dlds;

	if (x < X && y < Y) {
		dlds = grad[time*sets*total + set*total + ystart + (y*X+x)] * locd[time*sets*lsize + set*lsize + lstart + (y*X+x)];

		grad[time*sets*total + set*total + istart + (y*X+x)] += dlds;
		atomicAdd(meand + wsize*set + wstart + x, dlds);
	}
};