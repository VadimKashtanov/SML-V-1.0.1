#include "hip/hip_runtime.h"
#include "package/insts/gaussfiltre2d/head/gaussfiltre2d.cuh"

__global__
void gaussfiltre2d_use_th11(
	uint X, uint Y,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < X && y < Y) {
		var[time*total + ystart + y*X + x] = exp(-pow(var[time*total + istart + y*X + x] + weight[wstart + x],2));
	}
};