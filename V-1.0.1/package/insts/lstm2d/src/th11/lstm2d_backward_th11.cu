#include "hip/hip_runtime.h"
#include "package/insts/lstm2d/head/lstm2d.cuh"

/*
	Reecrire completement LSTM

	Mettre dans la classe des fonctions complexes

	Le méta-model genetic doit arriver lui meme avec `+`, `dot1d`, 'activ' ... a ça.

	On optimisera apres.

*/


/*			  =======
			  |     |
			  |		|
			  |	.W	|
			  |		|
			  |		|
			  =======
============= =======
|	.input	| | 	|	input@W
============= =======
				 +
			  =======
			  |	.U  |
			  |	    |
			  =======
	  ======= =======
	  |h[-1]| |		|  h[-1]@U
	  ======= =======
				 +
			  =======
			  |	.B	|
			  =======
*/

/*	We could use atomicAdd with 1 direct backward function

*/

/*
__global__  //ca veut dire que le kernel est position sur les cooredonne de l'input, et chaque kernel est associe a un pixel de l'input. Puis on backward on ligne verticale de .W
void lstm2d_backward_INPUT_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//	Backward grad(input)
	//	meand(.W) of f0,f1,f2,g0
	

	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	uint ipos = total*sets*time + total*set + istart + Bx*Ax + (y*Ax + x);

	//input = Ax*Ay, and the (x,y) pixel is in input. Then we backward .W and this pixel gradient
	if (x < Ax && y < Ay && pseudo_randomf(seed + ipos) > drop_rate) {	//if input[x] is droped, following will be *0

		float grad_input_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		//uint W = wsize*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		//uint vpos = total*sets*time + set*total + istart + x;
		float xvalue = var[ipos];

		float e;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos, outpos;	//wpos   = position du weight en question
												//epos,e_1pos,hpos = output `e` ou `h` (car output = `e` + `h`). e_1 est e[-1]
												//outpos = (y*Bx+k) juste pour calculer de quel pixel de Y nous prenon le locd (car on backward chaque colone de output mais les weights d'une meme ligne) 

		//	Backward W
		for (uint k=0; k < Bx; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y

			outpos = y*Bx + k;

			epos = out + outpos;
			e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
			hpos = out + Bx*Ay + outpos;

			e = var[epos];

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
			f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
			f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
			g0 = locd[locdpos + 3*Bx*Ay + outpos];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			//if time > 0:
			grad[e_1pos] += de * f0;		//we can't store only 4 locds, because how will we get de*f0 ?

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = wsize*set + wstart + 0*lineW + (k*Bx + y);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf0 * xvalue;
			atomicAdd(meand + wpos, dsf0 * xvalue);
			grad_input_compute += dsf0 * weight[wpos];

			//	f1
			wpos = wsize*set + wstart + 1*lineW + (k*Bx + y);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf1 * xvalue;
			atomicAdd(meand + wpos, dsf1 * xvalue);
			grad_input_compute += dsf1 * weight[wpos];

			//	f2
			wpos = wsize*set + wstart + 2*lineW + (k*Bx + y);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsf2 * xvalue;
			atomicAdd(meand + wpos, dsf2 * xvalue);
			grad_input_compute += dsf2 * weight[wpos];

			//	g0
			wpos = wsize*set + wstart + 3*lineW + (k*Bx + y);			//on met a jour que .W pas .U no .B
			//meand[wpos] += dsg0 * xvalue;
			atomicAdd(meand + wpos, dsg0 * xvalue);
			grad_input_compute += dsg0 * weight[wpos];
		}

		//	Backward input
		grad[ipos] += grad_input_compute;
		//atomicAdd(grad + vpos, grad_input_compute);
	}
}

__global__
void lstm2d_backward_H1_BIAS_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint x = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (x < Bx && y < Ay) {	//Only input is under drop. h is an output. It's values, could be droped, but in an other instruction

		float grad_H1_compute = 0;	//_INPUT_ et _H1_ ajoutent un gradient a input[x]

		uint h1pos = total*sets*(time-1) + set*total + istart + (y*Bx + x);	//h[-1] pos
		float h1val = var[h1pos];

		uint W = wsize*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		//float _grad;	//of h[t]

		//float chain_deriv;
		float e;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint wpos, epos, e_1pos, hpos, outpos;	//wpos   = position du weight en question
												//epos,e_1pos,hpos = output `e` ou `h` (car output = `e` + `h`). e_1 est e[-1]
												//outpos = (y*Bx+k) juste pour calculer de quel pixel de Y nous prenon le locd (car on backward chaque colone de output mais les weights d'une meme ligne) 

		for (uint k=0; k < Bx; k++) {	//[ w0 w1 w2 w3 ... wn]	une ligne du .W (la premiere par exemple)
										//car inp[x] est multiplice par `w[x*Bx + k] for k in Bx`  ou Bx==Y
										//en fait k est la colone de la matrice. la ligne est `y` du kernel
										//et le `x` du kernel determine le pixel `h[-1]` et la ligne dans .U 

			outpos = y*Bx + k;
			
			epos = out + outpos;
			e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
			hpos = out + Bx*Ay + outpos;

			dH = grad[hpos];

			f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
			f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
			f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
			g0 = locd[locdpos + 3*Bx*Ay + outpos];// * dH;

			de = grad[epos] + dH * f2;	//grad(e) += dH*f2

			grad[epos] = de;

			e = var[epos];

			//if time > 0:
			grad[e_1pos] += de * f0;

			dsf0 = de * var[e_1pos] * f0 * (1 - f0);
			dsf1 = de * g0 * f1 * (1 - f1);
			dsf2 = dH * e * f2 * (1 - f2);
			dsg0 = de * f1 * (1 - g0*g0);

			//	f0
			wpos = W + 0*lineW + _W + (x*Bx + k);					//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf0 * h1val;
			atomicAdd(meand + wpos, dsf0 * h1val);
			grad_H1_compute += dsf0 * weight[wpos];

			//	f1
			wpos = W + 1*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf1 * h1val;
			atomicAdd(meand + wpos, dsf1 * h1val);
			grad_H1_compute += dsf1 * weight[wpos];

			//	f2
			wpos = W + 2*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsf2 * h1val;
			atomicAdd(meand + wpos, dsf2 * h1val);
			grad_H1_compute += dsf2 * weight[wpos];
		
			//	g0
			wpos = W + 3*lineW + _W + (x*Bx + k);			//on met a jour que .U pas .W no .B
			//meand[wpos] += dsg0 * h1val;
			atomicAdd(meand + wpos, dsg0 * h1val);
			grad_H1_compute += dsg0 * weight[wpos];
		}

		//	Backward h[-1]
		grad[h1pos] += grad_H1_compute;
		//atomicAdd(grad + vpos, grad_input_compute);

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Bx,Ay>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//

		outpos = y*Bx + x;

		epos = out + outpos;
		e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
		hpos = out + Bx*Ay + outpos;

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
		f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
		f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
		g0 = locd[locdpos + 3*Bx*Ay + outpos];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[W + 0*lineW + _W + _U + (y*Bx + x)] += dsf0;

		//	f1
		meand[W + 1*lineW + _W + _U + (y*Bx + x)] += dsf1;

		//	f2
		meand[W + 2*lineW + _W + _U + (y*Bx + x)] += dsf2;

		//	g0
		meand[W + 3*lineW + _W + _U + (y*Bx + x)] += dsg0;
	}
};

__global__
void lstm2d_backward_BIAS_ONLY_th1x1(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	//
	//	h[-1] @ .U
	//

	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint x = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (x < Bx && y < Ay) {	//Only input is under drop. h is an output. It's values, could be droped, but in an other instruction

		uint W = wsize*set + wstart;
		uint out = total*sets*time + total*set + ystart;
		uint locdpos = lsize*sets*time + lsize*set + lstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		//float _grad;	//of h[t]

		//float chain_deriv;
		float e;
		float dH,f0,f1,f2,g0,de;
		float dsf0, dsf1, dsf2, dsg0;

		uint epos, e_1pos, hpos, outpos;	//wpos   = position du weight en question
												//epos,e_1pos,hpos = output `e` ou `h` (car output = `e` + `h`). e_1 est e[-1]
												//outpos = (y*Bx+k) juste pour calculer de quel pixel de Y nous prenon le locd (car on backward chaque colone de output mais les weights d'une meme ligne) 

		//  ============================================
		//	Backward .B
		//	Vu que la grille est de <<<Bx,Ay>>> on en profite car .B l'est aussi
		//	Au lieu de cree un autre fonction qui compute le gradient de .B, on le fait directe ici.	
		//
		
		outpos = y*Bx + x;

		epos = out + outpos;
		e_1pos = total*sets*(time-1) + total*set + ystart + outpos; //if l == 0 , e_1pos <= 0
		hpos = out + Bx*Ay + outpos;

		e = var[epos];

		dH = grad[hpos];

		f0 = locd[locdpos + 0*Bx*Ay + outpos];// * dH;
		f1 = locd[locdpos + 1*Bx*Ay + outpos];// * dH;
		f2 = locd[locdpos + 2*Bx*Ay + outpos];// * dH;
		g0 = locd[locdpos + 3*Bx*Ay + outpos];// * dH;

		de = grad[epos] + dH * f2;	//grad(e) += dH*f2
		grad[epos] = de;

		//if time > 0:
		grad[e_1pos] += de * f0;

		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);

		//	f0
		meand[W + 0*lineW + _W + _U + (y*Bx + x)] += dsf0;

		//	f1
		meand[W + 1*lineW + _W + _U + (y*Bx + x)] += dsf1;

		//	f2
		meand[W + 2*lineW + _W + _U + (y*Bx + x)] += dsf2;

		//	g0
		meand[W + 3*lineW + _W + _U + (y*Bx + x)] += dsg0;
	}
};
*/

__global__
void lstm2d_backward_th11(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint istart, uint ystart, uint wstart, uint lstart,
	uint total, uint wsize, uint lsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drate,
	uint sets)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	uint inp = total*sets*time + total*set + istart;
	uint W = wsize*set + wstart;
	uint out = total*sets*time + total*set + ystart;
	uint locdpos = lsize*sets*time + lsize*set + lstart;

	uint _W = Bx * Ax;
	uint _U = Bx * Bx;
	uint _B = Bx * Ay;
	
	uint lineW = _W + _U + _B;

	float f0=0,f1=0,f2=0,g0=0;

	float e, dH, de;
	float dsf0, dsf1, dsf2, dsg0;

	uint epos, e_1pos, hpos, vpos, wpos;
	
	if (x < Bx && y < Ay)
	{
		epos = out + (y*Bx + x);
		e_1pos = total*sets*(time-1) + total*set + ystart + (y*Bx + x); //if l == 0 , e_1pos <= 0
		hpos = out + Bx*Ay + (y*Bx + x);	//Bx*Ay is the space of `e`

		e = var[epos];
		dH = grad[hpos];
	
		f0 = locd[locdpos + 0*Bx*Ay + (y*Bx + x)] * dH;
		f1 = locd[locdpos + 1*Bx*Ay + (y*Bx + x)] * dH;
		f2 = locd[locdpos + 2*Bx*Ay + (y*Bx + x)] * dH;
		g0 = locd[locdpos + 3*Bx*Ay + (y*Bx + x)] * dH;
	
		de = grad[epos] + dH * f2;	//		#grad(e) += dH*f2
	
		grad[epos] = de;
	
		if (time > 0) {
			atomicAdd(&grad[e_1pos], de * f0);
		}
		dsf0 = de * var[e_1pos] * f0 * (1 - f0);
		dsf1 = de * g0 * f1 * (1 - f1);
		dsf2 = dH * e * f2 * (1 - f2);
		dsg0 = de * f1 * (1 - g0*g0);
	
		//// .W
		for (uint k=0; k < Ax; k++) {
			vpos = inp + y*Ax + k;

			if (pseudo_randomf(seed + vpos) > drate) {
				wpos = (k*Bx + x);
		
				//f0 += var[vpos]*w[W + wpos]
				atomicAdd(&grad[vpos], dsf0 * weight[W + 0*lineW + wpos]);
				atomicAdd(&meand[W + 0*lineW + wpos], dsf0 * var[vpos]);
		
				//f1 += var[vpos]*w[W + lineW + wpos]
				atomicAdd(&grad[vpos], dsf1 * weight[W + 1*lineW + wpos]);
				atomicAdd(&meand[W + 1*lineW + wpos], dsf1 * var[vpos]);
		
				//f2 += var[vpos]*w[W + 2*lineW + wpos]
				atomicAdd(&grad[vpos], dsf2 * weight[W + 2*lineW + wpos]);
				atomicAdd(&meand[W + 2*lineW + wpos], dsf2 * var[vpos]);
		
				//g0 += var[vpos]*w[W + 3*lineW + wpos]
				atomicAdd(&grad[vpos], dsg0 * weight[W + 3*lineW + wpos]);
				atomicAdd(&meand[W + 3*lineW + wpos], dsg0 * var[vpos]);
			}
		}
	
		//// .U
		if (time > 0) {
			for (uint k=0; k < Bx; k++) {
				//out == t
				//out - total*sets == sets*total*(l-1) + _set*total + istart
				vpos = sets*total*(time-1) + set*total + ystart + Bx*Ax + (y*Ax + k);// 	#h[-1][y][x]
				wpos = _W + (k*Bx + y);
	
				//f0 += var[vpos]*w[W + wpos]
				atomicAdd(&grad[vpos], dsf0 * weight[W + 0*lineW + wpos]);
				atomicAdd(&meand[W + 0*lineW + wpos], dsf0 * var[vpos]);
	
				//f1 += var[vpos]*w[W + lineW + wpos]
				atomicAdd(&grad[vpos], dsf1 * weight[W + 1*lineW + wpos]);
				atomicAdd(&meand[W + 1*lineW + wpos], dsf1 * var[vpos]);
	
				//f2 += var[vpos]*w[W + 2*lineW + wpos]
				atomicAdd(&grad[vpos], dsf2 * weight[W + 2*lineW + wpos]);
				atomicAdd(&meand[W + 2*lineW + wpos], dsf2 * var[vpos]);
	
				//g0 += var[vpos]*w[W + 3*lineW + wpos]
				atomicAdd(&grad[vpos], dsg0 * weight[W + 3*lineW + wpos]);
				atomicAdd(&meand[W + 3*lineW + wpos], dsg0 * var[vpos]);
			}
		}

		//// .B
		wpos = _W + _U + (y*Bx + x);
	
		atomicAdd(&meand[W + 0*lineW + wpos], dsf0);
		atomicAdd(&meand[W + 1*lineW + wpos], dsf1);
		atomicAdd(&meand[W + 2*lineW + wpos], dsf2);
		atomicAdd(&meand[W + 3*lineW + wpos], dsg0);
	}
}