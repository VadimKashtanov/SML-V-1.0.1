#include "hip/hip_runtime.h"
#include "package/insts/lstm2d/head/lstm2d.cuh"

__global__
void lstm2d_use_th11(
	uint Ax, uint Ay, uint Bx,
	uint time,
	uint total,
	uint istart, uint ystart, uint wstart,
	float * var, float * weight)
{
	uint x = threadIdx.x + blockIdx.x * blockDim.x;
	uint y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < Bx && y < Ay)
	{
		uint inp = total*time + istart;
		uint W = wstart;
		uint out = total*time + ystart;
		//uint locdpos = locds*time + locdstart;

		uint _W = Bx * Ax;
		uint _U = Bx * Bx;
		uint _B = Bx * Ay;

		uint lineW = _W + _U + _B;

		uint vpos, wpos;

		// f0,f1,f2 = logistic(x@W + h[-1]@U + B)
		// g0 	  = tanh 	(x@W + h[-1]@U + B)
		float f0=0,f1=0,f2=0,g0=0;

		float tmpt;

		// .W
		for (uint k=0; k < Ax; k++) {	//for all in INPUT
			//	Positions
			vpos = inp + (y*Ax + k);

			//
			wpos = k*Bx + y;

			tmpt = var[vpos];
			f0 += tmpt * weight[W + 0*lineW + wpos];
			f1 += tmpt * weight[W + 1*lineW + wpos];
			f2 += tmpt * weight[W + 2*lineW + wpos];
			g0 += tmpt * weight[W + 3*lineW + wpos];
		}

		// .U
		if (time > 0) {
			for (uint k=0; k < Bx; k++) {
				vpos = total*(time-1) + ystart + (Bx*Ay) + y*Bx + k;	///h[-1]
				wpos = _W + k*Bx + y;

				tmpt = var[vpos];
				f0 += tmpt * weight[W + 0*lineW + wpos];
				f1 += tmpt * weight[W + 1*lineW + wpos];
				f2 += tmpt * weight[W + 2*lineW + wpos];
				g0 += tmpt * weight[W + 3*lineW + wpos];
			}
		}

		// .B
		wpos = _W + _U + y*Bx + x;
		f0 += weight[W + 0*lineW + wpos];
		f1 += weight[W + 1*lineW + wpos];
		f2 += weight[W + 2*lineW + wpos];
		g0 += weight[W + 3*lineW + wpos];

		// activ(_sum)
		f0 = 1 / (1 + exp(-f0));
		f1 = 1 / (1 + exp(-f1));
		f2 = 1 / (1 + exp(-f2));
		g0 = tanh(g0);

		// e = f0 * e[-1] + f1 * g0
		// l - 1 have to be >= 0
		float e_1;
		if (time > 0) e_1 = var[total*(time-1) + ystart + y*Bx + x];
		else e_1 = 0;
		
		float e = f0*e_1 + f1*g0;
		float h = f2 * e;

		var[out + 0*Bx*Ay + y*Bx + x] = e;
		var[out + 1*Bx*Ay + y*Bx + x] = h;
	};
};