#include "hip/hip_runtime.h"
#include "package/insts/lstm2d/head/lstm2d.cuh"

#ifndef logistic
#define logistic(x) 1 / (1 + exp(-x))
#endif

//			   0  1  2     3      4      5       6        7 
//Arguments = [Ax,Ay,Bx, istart,ystart,wstart,locdstart, drate]

void lstm2d_check(uint * param) {
	if (param[0] == 0) raise(SIGINT);
	if (param[1] == 0) raise(SIGINT);
	if (param[2] == 0) raise(SIGINT);
	if (param[7] >100) raise(SIGINT);
};

void lstm2d_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;

	uint Ax=mdl->param[inst][0],		\
		 Ay=mdl->param[inst][1],		\
		 Bx=mdl->param[inst][2],		\
		 istart=mdl->param[inst][3],	\
		 ystart=mdl->param[inst][4],	\
		 wstart=mdl->param[inst][5];

	uint total = mdl->total;

	uint _W = Bx * Ax;
	uint _U = Bx * Bx;
	uint _B = Bx * Ay;

	uint lineW = _W + _U + _B;

	float * var = cpu->var;
	float * weight = mdl->weight;

	float f0,f1,f2,g0;
	float xval,hval;
	float e,e_1,h;

	for (uint x=0; x < Bx; x++) {
		for (uint y=0; y < Ay; y++) {
			//	Compute f0,f1,f2
			f0 = 0; f1 = 0; f2 = 0; g0 = 0;

			//x@.W
			for (uint k=0; k < Ax; k++) {
				xval = var[total*time + istart + (y*Ax + k)];

				f0 += weight[wstart + 0*lineW + (k*Bx + y)] * xval;
				f1 += weight[wstart + 1*lineW + (k*Bx + y)] * xval;
				f2 += weight[wstart + 2*lineW + (k*Bx + y)] * xval;
				g0 += weight[wstart + 3*lineW + (k*Bx + y)] * xval;
			}

			//h[-1]@.U
			if (time > 0) {
				for (uint k=0; k < Bx; k++) {
					hval = var[total*(time-1) + ystart + (Bx*Ay) + (y*Bx + k)];	//h[-1] is stored after e (of size Bx*Ay)

					f0 += weight[wstart + 0*lineW + _W + (k*Bx + y)] * hval;
					f1 += weight[wstart + 1*lineW + _W + (k*Bx + y)] * hval;
					f2 += weight[wstart + 2*lineW + _W + (k*Bx + y)] * hval;
					g0 += weight[wstart + 3*lineW + _W + (k*Bx + y)] * hval;
				}
			}

			f0 += weight[wstart + 0*lineW + _W + _U + (y*Bx + x)];
			f1 += weight[wstart + 1*lineW + _W + _U + (y*Bx + x)];
			f2 += weight[wstart + 2*lineW + _W + _U + (y*Bx + x)];
			g0 += weight[wstart + 3*lineW + _W + _U + (y*Bx + x)];

			f0 = 1 / (1 + exp(-f0));
			f1 = 1 / (1 + exp(-f1));
			f2 = 1 / (1 + exp(-f2));
			g0 = tanh(g0);

			if (time > 0) e_1 = var[total*(time-1) + ystart + (y*Bx + x)];
			else e_1 = 0;

			e = f0 * e_1 + f1 * g0;
			h = f2 * e;

			var[total*time + ystart + (y*Bx + x)] = e;
			var[total*time + ystart + Bx*Ay + (y*Bx + x)] = h;
		}
	}
};

void lstm2d_use(Use_t * use, uint inst, uint time) {
	lstm2d_use_call_mode_th11(use, inst, time);
};

void lstm2d_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm2d_forward_call_mode_th11(train, inst, time, start_seed);
};

void lstm2d_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	lstm2d_backward_call_mode_th11(train, inst, time, start_seed);
};