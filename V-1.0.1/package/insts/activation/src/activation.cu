#include "hip/hip_runtime.h"
#include "package/insts/activation/head/activation.cuh"

void activation_check(uint * param) {
	//>0 <==> >= 1
	if (param[0] == 0) raise(SIGINT);
	if (param[1] >= 4) raise(SIGINT);
};

void activation_cpu(Cpu_t * cpu, uint inst, uint time) {
	Mdl_t * mdl = cpu->mdl;
	uint total = mdl->total;

	uint _len=mdl->param[inst][0],			\
		 activ=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3];

	float * var = cpu->var;
	
	for (uint i=0; i < _len; i++) {
		float value = var[time*total + istart + i];

		if (activ == 0) value = 1 / (1 + exp(-value));
		else if (activ == 1) activ = tanh(value);
		else if (activ == 2) activ = exp(-value*value);
		else if (activ == 3) activ *= (activ > 0);

		var[time*total + ystart + i] = value;
	}
};

void activation_use(Use_t * use, uint inst, uint time) {
	activation_use_call_mode_th11(use, inst, time);
};

void activation_forward(Train_t * train, uint inst, uint time, uint start_seed) {
	activation_forward_call_mode_th11(train, inst, time, start_seed);
};

void activation_backward(Train_t * train, uint inst, uint time, uint start_seed) {
	activation_backward_call_mode_th11(train, inst, time, start_seed);
};