#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void activation_forward_th11(
	uint _len,
	uint activ,
	uint time,
	uint istart, uint ystart, uint locdstart,
	uint total, uint locdsize,
	float * var, float * locd,
	uint sets)
{
	uint i = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = blockIdx.y;

	if (i < _len) {
		float value = var[time*sets*total + set*total + istart + i];
		float __locd;

		if (activ == 0) {
			value = 1 / (1 + exp(-value));
			__locd = value*(1 - value);	//f'(x) = f(x)(1 - f(x))
		} else if (activ == 1) {
			value = tanh(value);
			__locd = 1 - value*value;	//f'(x) = 1 - tanh(x)^2
		} else if (activ == 2) {
			__locd = value;
			value = exp(-value*value);
			__locd = -2*__locd*value;	//f'(x) = -2x*e^(-x^2)
		} else  if (activ == 3) {
			__locd = (value > 0);
			value = value*__locd;
		} else if (activ == 4) {
			__locd = 1;
		}

		var[time*sets*total + set*total + ystart + i] = value;		//same assembler than putting it in if/else structure
		locd[time*sets*locdsize + set*locdsize + locdstart + i] = __locd;
	}
};
