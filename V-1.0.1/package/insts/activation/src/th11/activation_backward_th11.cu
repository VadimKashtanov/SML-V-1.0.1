#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void activation_backward_th11(
	uint _len,
	uint activ,
	uint time,
	uint istart, uint ystart, uint locdstart,
	uint total, uint locdsize,
	float * var, float * locd, float * grad,
	uint sets)
{
	uint i = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = blockIdx.y;

	if (i < _len) {
		float dlds = grad[time*sets*total + set*total + ystart + i] * locd[time*sets*locdsize + set*locdsize + locdstart + i];

		var[time*sets*total + set*total + istart + i] += dlds;
	}
};