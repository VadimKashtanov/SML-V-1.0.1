#include "hip/hip_runtime.h"
#include "package/insts/dot1d/head/dot1d.cuh"

__global__
void activation_use_th11(
	uint _len,
	uint activ,
	uint time,
	uint total,
	uint istart, uint ystart,
	float * var)
{
	uint i = threadIdx.x + blockIdx.x*blockDim.x;

	if (i < _len) {
		float value = var[time*total + istart + i];

		if (activ == 0) value = 1 / (1 + exp(-value));
		else if (activ == 1) activ = tanh(value);
		else if (activ == 2) activ = exp(-value*value);
		else if (activ == 3) activ *= (activ > 0);

		var[time*total + ystart + i] = value;
	}
};