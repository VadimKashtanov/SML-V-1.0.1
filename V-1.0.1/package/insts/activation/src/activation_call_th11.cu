#include "package/insts/activation/head/activation.cuh"

void activation_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;

	uint _len=mdl->param[inst][0],			\
		 activ=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3];

	activation_use_th11<<<dim3(KERN_DIV(_len,32)),dim3(32)>>>(
		_len,
		activ,
		time,
		mdl->total,
		istart, ystart,
		use->var_d);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//======================== Train_t =======================

//-------------------------- forward ---------------------

void activation_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint _len=mdl->param[inst][0],			\
		 activ=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3],		\
		 lstart=mdl->param[inst][4];

	activation_forward_th11<<<dim3(KERN_DIV(_len,16),train->sets),dim3(16,1)>>>(
		_len,
		activ,
		time,
		istart, ystart, lstart,
		train->mdl->total, train->mdl->locds,
		train->_var, train->_locd,
		train->sets);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//-------------------------- backward ---------------------

void activation_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint _len=mdl->param[inst][0],			\
		 activ=mdl->param[inst][1],			\
		 istart=mdl->param[inst][2],	\
		 ystart=mdl->param[inst][3],		\
		 lstart=mdl->param[inst][4];

	activation_backward_th11<<<dim3(KERN_DIV(_len,16),train->sets),dim3(16,1)>>>(
		_len,
		activ,
		time,
		istart, ystart, lstart,
		mdl->total, mdl->locds,
		train->_var, train->_locd, train->_grad,
		train->sets);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};