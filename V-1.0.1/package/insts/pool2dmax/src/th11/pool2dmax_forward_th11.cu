#include "hip/hip_runtime.h"
#include "package/insts/pool2dmax/head/pool2dmax.cuh"

__global__
void pool2dmax_forward_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,						
	uint time,
	uint total, uint locds,
	uint istart, uint ystart, uint locdstart,
	uint sets,
	float * var, float * locd)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _set = blockIdx.z;

	if (_Yx < Yx && _Yy < Yy) {
		float _max = var[time*sets*total + _set*total + istart + (_Yy*Ypool + 0)*Ax + (_Yx*Xpool + 0)];
		
		uint index = 0;

		float _compare;

		for (uint _y=0; _y < Ypool; _y++) {
			for (uint _x=0; _x < Xpool; _x++) {
				_compare = var[time*sets*total + _set*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)];
				if (_compare > _max) {
					_max = _compare;
					index = _y*Xpool + _x;
				}
			}
		}

		var[time*sets*total + _set*total + ystart + _Yy*Yx + _Yx] = _max;
		locd[time*sets*locds + _set*locds + locdstart + _Yy*Yx + _Yx] = (float)index;
	};
}