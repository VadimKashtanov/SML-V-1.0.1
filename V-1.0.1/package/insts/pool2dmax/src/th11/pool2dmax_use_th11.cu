#include "hip/hip_runtime.h"
#include "package/insts/pool2dmax/head/pool2dmax.cuh"

__global__
void pool2dmax_use_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,
	uint time,
	uint total,
	uint istart, uint ystart,
	float * var)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;

	if (_Yx < Yx && _Yy < Yy) {
		float _max = var[time*total + istart + (_Yy*Ypool + 0)*Ax + (_Yx*Xpool + 0)];
		float _compare;

		for (uint _y=0; _y < Ypool; _y++) {
			for (uint _x=0; _x < Xpool; _x++) {
				_compare = var[time*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)];
				if (_compare > _max)
					_max = _compare;
			}
		}

		var[time*total + ystart + _Yy*Yx + _Yx] = _max;
	};
};