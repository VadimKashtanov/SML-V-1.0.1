#include "hip/hip_runtime.h"
#include "package/insts/pool2dmax/head/pool2dmax.cuh"

__global__
void pool2dmax_backward_th1x1(
	uint Yx, uint Yy,
	uint Ax, uint Ay, uint Xpool, uint Ypool,
	uint time,
	uint total, uint locds,
	uint istart, uint ystart, uint locdstart,
	uint sets,
	float * var, float * locd,
	float * grad)
{
	uint _Yx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _Yy = threadIdx.y + blockIdx.y * blockDim.y;
	uint _set = blockIdx.z;

	if (_Yx < Yx && _Yy < Yy) {
		float dl_dpoolmax = grad[time*sets*total + _set*total + ystart + _Yy*Yx + _Yx];

		uint locd_val = (uint)locd[time*sets*locds + _set*locds + locdstart + _Yy*Yx + _Yx];

		uint _x = locd_val % Xpool;			//index en x du maximum du block
		uint _y = (locd_val-_x) / Xpool;	//et en y

		grad[time*sets*total + _set*total + istart + (_Yy*Ypool + _y)*Ax + (_Yx*Xpool + _x)] += dl_dpoolmax;
	}
}