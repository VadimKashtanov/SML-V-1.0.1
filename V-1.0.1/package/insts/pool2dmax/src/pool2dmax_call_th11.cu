#include "package/insts/pool2dmax/head/pool2dmax.cuh"

void pool2dmax_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	//
	Mdl_t * mdl = use->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	//uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];
	//uint locdstart = param[6];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2dmax_use_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32)), dim3(32,32)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,
		time,
		total,
		istart, ystart,
		use->var_d);

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}

void pool2dmax_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];
	uint locdstart = param[6];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2dmax_forward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), train->sets), dim3(32,32, 1)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,				
		time,
		total, locds,
		istart, ystart, locdstart,
		train->sets,
		train->_var, train->_locd);
	
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}

void pool2dmax_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	//
	Mdl_t * mdl = train->mdl;
	uint total = mdl->total;
	//uint weights = mdl->weights;
	uint locds = mdl->locds;

	//
	uint * param = mdl->param[ inst ];
	uint Ax = param[0];
	uint Ay = param[1];
	uint Xpool = param[2];
	uint Ypool = param[3];
	uint istart = param[4];
	uint ystart = param[5];
	uint locdstart = param[6];

	uint Yx = Ax / Xpool;
	uint Yy = Ay / Ypool;

	pool2dmax_backward_th1x1<<<dim3(KERN_DIV(Yx,32), KERN_DIV(Yy,32), train->sets), dim3(32,32, 1)>>>(
		Yx, Yy,
		Ax, Ay,
		Xpool, Ypool,
		time,
		total, locds,
		istart, ystart, locdstart,
		train->sets,
		train->_var, train->_locd,
		train->_grad);

	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
}