#include "hip/hip_runtime.h"
#include "package/scores/crossentropy/head/crossentropy.cuh"

//=================================================================================================
//===================================== dLOSS(g,w)/dg =============================================
//=================================================================================================

static __global__ void opti_kernel_ce_dloss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs,
	uint sets)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;

		float __out = var[pos];

		if (__out == 0) {
			printf("\033[91mThere is 0 values with crossentropy. Division by 0 doesn't exists. Make sure last instruction produce >0 values only.\033[0m\n");
			assert(0);
		}

		grad[pos] = -output[line*outs + out]/__out;
	};
};

void CROSSENTROPY_dloss(Opti_t * opti) {
	Train_t * train = opti->train;

	uint outpos = train->mdl->total - train->mdl->outputs;

	opti_kernel_ce_dloss<<<dim3(KERN_DIV(train->mdl->outputs, 32), KERN_DIV(train->data->lines, 32), train->sets),dim3(32, 32, 1)>>>(
		train->_grad, train->_var, train->data->output_d,
		train->mdl->total, outpos, train->data->lines, train->data->outputs,
		train->sets
	);
	SAFE_CUDA(hipPeekAtLastError());
};

//=================================================================================================
//====================================== LOSS(g,w) ================================================
//=================================================================================================

static __global__ void opti_kernel_ce_loss(
	float * grad, float * var, float * output,
	uint total, uint ostart, uint lines, uint outs,
	uint sets)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint line = threadIdx.y + blockIdx.y * blockDim.y;
	uint set = blockIdx.z;

	if (out < outs && line < lines)
	{
		uint pos = line*sets*total + set*total + ostart + out;
		float g = var[pos];
		float w = output[line*outs + out];

		if (g == 0) {
			//on peut pas utiliser ERR() dans les fonctions __host__
			printf("\033[91mThere is 0 values with crossentropy. log(0) doesn't exists. Make sure last instruction produce >0 values only.\033[0m\n");
			assert(0);
		}

		grad[pos] = -w*log(g);
	};
};

static __global__ void opti_kernel_sum_scores_over_lines(
	float * grad, float * var, float * output,
	float * score_one_line_d,
	uint total, uint lines, uint sets, uint ostart, uint outs)
{
	uint out = threadIdx.x + blockIdx.x * blockDim.x;
	uint set = blockIdx.y;

	if (out < outs)
	{
		//uint pos;
		float _sum_of_lines = 0;
		for (uint l=0; l < lines; l++) {
			_sum_of_lines += grad[l*sets*total + set*total + ostart + out];
		}
		score_one_line_d[set*total + out] = _sum_of_lines / lines;
	};
};

static __global__ void opti_kernel_sum_scores_over_outputs(
	float * score_one_line_d, float * scores,
	uint total, uint sets, uint ostart, uint outs)
{
	uint set = blockIdx.x;

	uint start = set*total + 0;
	float _sum_of_outs = 0;
	for (uint o=0; o < outs; o++) {
		_sum_of_outs += score_one_line_d[start];
		start++;
	}

	scores[set] = _sum_of_outs / outs;
};

void CROSSENTROPY_loss(Opti_t * opti) {
	Train_t * train = opti->train;
	Mdl_t * mdl = train->mdl;

	uint outs = mdl->outputs;
	uint lines = train->data->lines;
	uint sets = train->sets;
	uint out_start = mdl->total - outs;

	//======================================================================

	//						compute score

	opti_kernel_ce_loss<<<dim3(KERN_DIV(outs, 32), KERN_DIV(lines, 32), sets),dim3(32,32,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		mdl->total, out_start, lines, outs,
		sets);
	SAFE_CUDA(hipPeekAtLastError());

	//======================================================================
	//======================================================================

	//				sum over lines (only outputs)

	float * score_one_line_d;
	SAFE_CUDA(hipMalloc((void**)&score_one_line_d, sizeof(float) * sets * outs));	//all lines are sumed in one (only outputs)

	opti_kernel_sum_scores_over_lines<<<dim3(KERN_DIV(outs, 16), sets),dim3(16,1)>>>(
		train->_grad, train->_var, train->data->output_d,
		score_one_line_d,
		mdl->total, lines, sets, out_start, outs);
	SAFE_CUDA(hipPeekAtLastError());

	//======================================================================
	//======================================================================

	//		sum of output pixels

	opti_kernel_sum_scores_over_outputs<<<dim3(sets),dim3(1)>>>(
		score_one_line_d, opti->set_score_d,
		mdl->total, sets, out_start, outs);
	SAFE_CUDA(hipPeekAtLastError());

	SAFE_CUDA(hipFree(score_one_line_d));
};