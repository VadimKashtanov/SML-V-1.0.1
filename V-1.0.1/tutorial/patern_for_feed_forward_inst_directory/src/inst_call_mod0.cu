#include "package/insts/inst/head/inst.cuh"

void inst_use_call_mode_th11(Use_t * use, uint inst, uint time) {
	Mdl_t * mdl = use->mdl;

	uint param0=mdl->param[inst][0],\
		 param1=mdl->param[inst][1];

	inst_use_mod0<<<dim3(KERN_DIV(Yx,32)),dim3(32)>>>(
		param0, param1,
		time,
		mdl->total,
		input_start, ystart, wstart,
		use->var_d, use->weight_d);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//======================== Train_t =======================

//-------------------------- forward ---------------------

void dot1d_forward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint param0=mdl->param[inst][0],\
		 param1=mdl->param[inst][1];

	float param1_100 = param1/100;

	inst_forward_mod0<<<dim3(KERN_DIV(param1,16),train->sets),dim3(16,1)>>>(
		param0, param1,
		time,
		input_start, ystart, wstart, locdstart,
		train->mdl->total, train->mdl->weights, train->mdl->locds,
		train->_var, train->_weight, train->_locd,
		inst*start_seed, drop_rate,
		train->sets);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//-------------------------- backward ---------------------

void dot1d_backward_call_mode_th11(Train_t * train, uint inst, uint time, uint start_seed) {
	Mdl_t * mdl = train->mdl;

	uint param0=mdl->param[inst][0],\
		 param1=mdl->param[inst][1];

	float param1_100 = param1/100;

	inst_backward_mod0<<<dim3(KERN_DIV(param1,16),train->sets),dim3(16,1)>>>(
		param0, param1,
		time,
		input_start, ystart, wstart, locdstart,
		mdl->total, mdl->weights, mdl->locds,
		train->_var, train->_weight, train->_locd, train->_grad, train->_meand,
		inst*start_seed, drop_rate,
		train->sets);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};