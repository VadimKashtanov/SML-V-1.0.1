#include "hip/hip_runtime.h"
#include "package/insts/inst/head/inst.cuh"

__global__
void inst_backward_mod0(
	uint param0, uint parram1,
	uint time,
	uint input_start, uint ystart, uint wstart, uint locdstart,
	uint total, uint wsize, uint locdsize,
	float * var, float * weight, float * locd, float * grad, float * meand,
	uint seed, float drop_rate,
	uint sets)
{
	/*	Kernel coordinates	*/
	uint _Yx = threadIdx.x + blockIdx.x*blockDim.x, \
		 set = blockIdx.y;

	if (_Yx < parram1) {
		
	}
};