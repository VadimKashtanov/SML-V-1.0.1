#include "kernel/head/optis.cuh"

/*		-------------    Build  ------------ */
Opti_t * opti_mk(Train_t * train, uint score_algo, uint opti_algo) {
	if (score_algo >= SCORES)
		ERR("Score number %i doesn't exists. Max is %i", score_algo, SCORES - 1)
	if (opti_algo >= OPTIS)
		ERR("Opti number %i doesn't exists. Max is %i", opti_algo, OPTIS - 1)

	Opti_t * ret = (Opti_t*)malloc(sizeof(Opti_t));
	
	ret->train = train;

	//	Cpu ram arrays
	ret->set_score = (float*)malloc(sizeof(float) * train->sets);
	ret->set_rank = (uint*)malloc(sizeof(uint) * train->sets);

	ret->podium = (uint*)malloc(sizeof(uint) * train->sets);

	//	Gpu vram arrays
	SAFE_CUDA(hipMalloc((void**)&ret->set_score_d, sizeof(float) * train->sets));
	SAFE_CUDA(hipMalloc((void**)&ret->set_rank_d, sizeof(uint) * train->sets));

	//	Algorithms
	ret->score_algo = score_algo;
	ret->opti_algo = opti_algo;

	ret->score_space = OPTI_SCORE_SPACE_MK_ARRAY[score_algo](ret);
	ret->opti_space = OPTI_OPTI_SPACE_MK_ARRAY[opti_algo](ret);

	return ret;
};

/*		-------------    Free structure  ------------ */
void opti_free(Opti_t * opti) {
	free(opti->set_score);
	free(opti->set_rank);
	free(opti->podium);

	SAFE_CUDA(hipFree(opti->set_score_d));
	SAFE_CUDA(hipFree(opti->set_rank_d));

	OPTI_FREE_SCORE_ARRAY[opti->score_algo](opti);
	OPTI_FREE_OPTI_ARRAY[opti->opti_algo](opti);

	free(opti);
};