#include "kernel/head/testpackage.cuh"

float * load_float_array(uint len, FILE * fp) {
	float * ret = (float*)malloc(sizeof(float) * len);
	fread(ret, sizeof(float), len, fp);
	return ret;
};

void read_123(FILE * fp) {
	//Read next `uint` and check if it's 123.
	//if not, there is a probleme in reading. Maybe read to much or not enougth

	uint _tmp;
	fread(&_tmp, sizeof(uint), 1, fp);
	if (_tmp != 123)
		ERR("Expected an 123 number, but get : %i", 123);
};

//=====================================================================

static bool compare_arrays(float * cpu0, float * cpu1, uint count)
{
	for (uint i=0; i < count; i++) {
		if (compare_floats(cpu0[i], cpu1[i], 0.0001) != true) {
			return false;
		}
	}
	return true;
};

bool test_package_compare_cpu_and_gpu(float * cpu0, float * gpu_d, uint count)
{
	float * cpu = (float*)malloc(sizeof(float) * count);
	SAFE_CUDA(hipMemcpy(cpu, gpu_d, sizeof(float) * count, hipMemcpyDeviceToHost));
	bool ret = compare_arrays(cpu0, cpu, count);
	free(cpu);
	return ret;
};

bool test_package_compare_cpu_and_cpu(float * cpu0, float * cpu1, uint count)
{
	return compare_arrays(cpu0, cpu1, count);
};

//==========================================================================

Data_t * load_test_data(FILE * fp)
{
	uint batchs, lines, inputs, outputs;

	fread(&batchs, sizeof(uint), 1, fp);
	fread(&lines, sizeof(uint), 1, fp);
	fread(&inputs, sizeof(uint), 1, fp);
	fread(&outputs, sizeof(uint), 1, fp);

	Data_t * ret = data_load(batchs, inputs, outputs, lines);

	data_cudmalloc(ret);

	fread(ret->input, sizeof(float), lines*inputs, fp);
	fread(ret->output, sizeof(float), lines*outputs, fp);

	data_cudamemcpy(ret);

	return ret;
};