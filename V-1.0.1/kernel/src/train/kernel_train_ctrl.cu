#include "hip/hip_runtime.h"
#include "kernel/head/train.cuh"

static __global__ void kernel_random_weights(uint rnd_seed, uint weights, float * _weight) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights) {
		//printf("%i %i %f\n", rnd_seed + pos, pos, pseudo_randomf_minus1_1(rnd_seed + pos));
		//pseudo_randomf()
		_weight[pos] = pseudo_randomf_minus1_1(rnd_seed + pos);
	}
};

void train_random_weights(Train_t * train, uint rnd_seed) {
	kernel_random_weights<<<dim3(KERN_DIV(train->mdl->weights,32), train->sets), dim3(32,1)>>>(
		rnd_seed, train->mdl->weights, train->_weight);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());
};

//	---------------------------------------------------------------------------------------------
//	---------------------------------------------------------------------------------------------

static __global__ void kernel_random_weights_from_mdl(uint rnd_seed, uint weights, float * _weight, float * mdl_weight_d) {
	uint wid = threadIdx.x + blockIdx.x*blockDim.x;
	uint set = threadIdx.y + blockIdx.y*blockDim.y;
	uint pos = set*weights + wid;

	if (wid < weights) {
		//_weight[pos] = mdl_weight_d[wid] + 0.02*(pseudo_randomf(rnd_seed + pos) - 0.5);//0.01*(2*(rnd()-0.5)))
		_weight[pos] = mdl_weight_d[wid] + 0.05*pseudo_randomf_minus1_1(rnd_seed + pos);
	}
};

void train_random_weights_from_mdl(Train_t * train, uint rnd_seed) {
	float * mdl_weights_d;
	SAFE_CUDA(hipMalloc((void**)&mdl_weights_d, sizeof(float)*train->mdl->weights));
	SAFE_CUDA(hipMemcpy(mdl_weights_d, train->mdl->weight, sizeof(float)*train->mdl->weights, hipMemcpyHostToDevice));

	kernel_random_weights_from_mdl<<<dim3(KERN_DIV(train->mdl->weights, 32), train->sets),dim3(32,1)>>>(
		rnd_seed, train->mdl->weights, train->_weight, mdl_weights_d);
	hipDeviceSynchronize();
	SAFE_CUDA(hipPeekAtLastError());

	SAFE_CUDA(hipFree(mdl_weights_d));
};

//	---------------------------------------------------------------------------------------------
//	---------------------------------------------------------------------------------------------

void train_cpy_ws_to_mdl(Train_t * train, uint set) {
	SAFE_CUDA(hipMemcpy(
		train->mdl->weight, train->_weight + set*train->mdl->weights,
		sizeof(float)*train->mdl->weights, hipMemcpyDeviceToHost));
};

//	---------------------------------------------------------------------------------------------
//	---------------------------------------------------------------------------------------------

Train_t * extract_to_new_train(Train_t * old, uint amount, uint * set_id) {
	Train_t * new_train = mk_train(old->mdl, old->data, amount);
	
	uint ws = old->mdl->weights;

	for (uint s=0; s < amount; s++)
		SAFE_CUDA(hipMemcpy(new_train->_weight + s*ws, old->_weight + set_id[s]*ws, sizeof(float)*ws, hipMemcpyDeviceToDevice))

	return new_train;
};

//	---------------------------------------------------------------------------------------------
//	---------------------------------------------------------------------------------------------

static __global__ void kernel_set_input(float * _var, float * _input, uint total, uint sets, uint inputs, uint lines) {
	uint _inp = threadIdx.x + blockIdx.x * blockDim.x,	\
		 line = threadIdx.y + blockIdx.y * blockDim.y,	\
		 set = blockIdx.z;

	if (_inp < inputs && line < lines) {
		_var[line*sets*total + set*total + _inp] = _input[line*inputs + _inp];
	}
};

void train_set_input(Train_t * train) {
	kernel_set_input<<<dim3(KERN_DIV(train->mdl->inputs,32), KERN_DIV(train->data->lines,32), train->sets),dim3(32,32,1)>>>(
		train->_var, train->data->input_d, train->mdl->total, train->sets, train->mdl->inputs, train->data->lines);
	SAFE_CUDA(hipPeekAtLastError());
};

//	---------------------------------------------------------------------------------------------
//	---------------------------------------------------------------------------------------------

void train_null_grad_meand(Train_t * train) {
	SAFE_CUDA(hipMemset(train->_meand, 0, sizeof(float) * train->sets * train->mdl->weights))
	SAFE_CUDA(hipMemset(train->_grad, 0, sizeof(float) * train->sets * train->data->lines * train->mdl->total))
};

void train_forward(Train_t * train, uint start_seed) {
	for (uint t=0; t < train->data->lines; t++) {
		for (uint i=0; i < train->mdl->insts; i++) {
			INST_FORWARD[train->mdl->id[i]](train, i, t, start_seed);
		}
	}
};

void train_backward(Train_t * train, uint start_seed) {
	for (int t=train->data->lines-1; t >= 0; t--) {
		for (int i=train->mdl->insts-1; i >= 0; i--) {
			INST_BACKWARD[train->mdl->id[i]](train, i, t, start_seed);
		}
	}
};
