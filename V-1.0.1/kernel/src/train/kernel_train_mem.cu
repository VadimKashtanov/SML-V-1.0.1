#include "kernel/head/train.cuh"

Train_t* mk_train(Mdl_t * mdl, Data_t * data, uint sets)
{
	Train_t * ret = (Train_t*)malloc(sizeof(Train_t));

	ret->mdl = mdl;
	ret->data = data;
	ret->sets = sets;

	uint lines = data->lines;

	SAFE_CUDA(hipMalloc((void**)&ret->_weight, sizeof(float) * (mdl->weights*sets)));
	SAFE_CUDA(hipMalloc((void**)&ret->_var, sizeof(float) * (mdl->total*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_locd, sizeof(float) * (mdl->locds*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_grad, sizeof(float) * (mdl->total*sets*lines)));
	SAFE_CUDA(hipMalloc((void**)&ret->_meand, sizeof(float) * (mdl->weights*sets)));

	return ret;
};

void train_free(Train_t * train) {
	SAFE_CUDA(hipFree(train->_weight));
	SAFE_CUDA(hipFree(train->_var));
	SAFE_CUDA(hipFree(train->_locd));
	SAFE_CUDA(hipFree(train->_grad));
	SAFE_CUDA(hipFree(train->_meand));

	free(train);
};

