#include "kernel/head/train.cuh"

static void vsep_print(Separators_t * sep, float * arr0, uint sets, uint lines, uint total)
{
	int lbl;
	uint pos;

	for (uint l=0; l < lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint s=0; s < sets; s++) {
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
			printf("Set #%i ============= \n", s);
			for (uint i=0; i < total; i++) {
				lbl = find_sep(sep, i);

				if (lbl != -1) {
					printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
					printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
					printf("|| (%i) %s\n", i, sep->labels[lbl]);
				}

				pos = l*total*sets + s*total + i;
				
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
				printf("|| %i |  \033[93m %f \033[0m \n", i, arr0[pos]);
			}
		}
	}
};

static void wsep_print(Separators_t * sep, float * arr0, uint sets, uint weights)
{
	int lbl;
	uint pos;

	for (uint s=0; s < sets; s++) {
		printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
		printf("Set #%i ============= \n", s);
		for (uint i=0; i < weights; i++) {
			lbl = find_sep(sep, i);

			if (lbl != -1) {
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				printf("|| (%i) %s\n", i, sep->labels[lbl]);
			}

			pos = weights*s + i;
				
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
			printf("|| %i |  \033[93m %f \033[0m \n", i, arr0[pos]);
		}
	}
};

static void lsep_print(Separators_t * sep, float * arr0, uint sets, uint lines, uint locds)
{
	int lbl;
	uint pos;

	for (uint l=0; l < lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint s=0; s < sets; s++) {
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
			printf("Set #%i ============= \n", s);
			for (uint i=0; i < locds; i++) {
				lbl = find_sep(sep, i);

				if (lbl != -1) {
					printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
					printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
					printf("|| (%i) %s\n", i, sep->labels[lbl]);
				}

				pos = l*locds*sets + s*locds + i;
				
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
				printf("|| %i |  \033[93m %f \033[0m \n", i, arr0[pos]);
			}
		}
	}
};

//	---------------------------------------------------------------------------------------------

void train_print_weights(Train_t * train) {
	uint sets = train->sets;
	uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	//uint lines = train->data->lines;

	float * arr1 = (float*)malloc(sizeof(float) * (sets * weights));
	SAFE_CUDA(hipMemcpy(arr1, train->_weight, sizeof(float) * (sets * weights), hipMemcpyDeviceToHost));
	wsep_print(train->mdl->wsep, arr1, sets, weights);
	free(arr1);
};

void train_print_vars(Train_t * train) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * arr1 = (float*)malloc(sizeof(float) * (sets * lines * total));
	SAFE_CUDA(hipMemcpy(arr1, train->_var, sizeof(float) * (sets * lines * total), hipMemcpyDeviceToHost));
	vsep_print(train->mdl->vsep, arr1, sets, lines, total);
	free(arr1);
};

void train_print_locds(Train_t * train) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * arr1 = (float*)malloc(sizeof(float) * (sets * lines * locds));
	SAFE_CUDA(hipMemcpy(arr1, train->_locd, sizeof(float) * (sets * lines * locds), hipMemcpyDeviceToHost));
	lsep_print(train->mdl->lsep, arr1, sets, lines, locds);
	free(arr1);
};

void train_print_grads(Train_t * train) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * arr1 = (float*)malloc(sizeof(float) * (sets * lines * total));
	SAFE_CUDA(hipMemcpy(arr1, train->_grad, sizeof(float) * (sets * lines * total), hipMemcpyDeviceToHost));
	vsep_print(train->mdl->vsep, arr1, sets, lines, total);
	free(arr1);
};

void train_print_meands(Train_t * train) {
	uint sets = train->sets;
	uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	//uint lines = train->data->lines;

	float * arr1 = (float*)malloc(sizeof(float) * (sets * weights));
	SAFE_CUDA(hipMemcpy(arr1, train->_meand, sizeof(float) * (sets * weights), hipMemcpyDeviceToHost));
	wsep_print(train->mdl->wsep, arr1, sets, weights);
	free(arr1);
};

//
void train_print_all(Train_t * train) {
	printf(" ============== WEIGHTS ==============\n");
	train_print_weights(train);
	printf(" ============== VARS ==============\n");
	train_print_vars(train);
	printf(" ============== LOCDS ==============\n");
	train_print_locds(train);
	printf(" ============== GRADS ==============\n");
	train_print_grads(train);
	printf(" ============== MEANDS ==============\n");
	train_print_meands(train);
};
