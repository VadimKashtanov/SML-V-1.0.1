#include "kernel/head/train.cuh"

static void vsep_compare_cpu_cpu(Separators_t * sep, float * arr0, float * arr1, uint sets, uint lines, uint total)
{
	int lbl;
	uint pos;

	for (uint l=0; l < lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint s=0; s < sets; s++) {
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
			printf("Set #%i ============= \n", s);
			for (uint i=0; i < total; i++) {
				lbl = find_sep(sep, i);

				if (lbl != -1) {
					printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
					printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
					printf("|| (%i) %s\n", i, sep->labels[lbl]);
				}

				pos = l*total*sets + s*total + i;
				
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
				if (compare_floats(arr0[pos], arr1[pos], COMPARE_DEEPH)) 
					printf("|| %i |  \033[42m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
				else
					printf("|| %i |  \033[41m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
			}
		}
	}
};

static void wsep_compare_cpu_cpu(Separators_t * sep, float * arr0, float * arr1, uint sets, uint weights)
{
	int lbl;
	uint pos;

	for (uint s=0; s < sets; s++) {
		printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
		printf("Set #%i ============= \n", s);
		for (uint i=0; i < weights; i++) {
			lbl = find_sep(sep, i);

			if (lbl != -1) {
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				printf("|| (%i) %s\n", i, sep->labels[lbl]);
			}

			pos = weights*s + i;
				
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
			if (compare_floats(arr0[pos], arr1[pos], COMPARE_DEEPH)) 
				printf("|| %i |  \033[42m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
			else
				printf("|| %i |  \033[41m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
		}
	}
};

static void lsep_compare_cpu_cpu(Separators_t * sep, float * arr0, float * arr1, uint sets, uint lines, uint locds)
{
	int lbl;
	uint pos;

	for (uint l=0; l < lines; l++) {
		printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));
		printf("Line = %i ################### \n", l);
		for (uint s=0; s < sets; s++) {
			printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
			printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
			printf("Set #%i ============= \n", s);
			for (uint i=0; i < locds; i++) {
				lbl = find_sep(sep, i);

				if (lbl != -1) {
					printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
					printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
					printf("|| (%i) %s\n", i, sep->labels[lbl]);
				}

				pos = l*locds*sets + s*locds + i;
				
				printf("\033[%im||\033[0m", (l % 2 ? 92 : 91));	// '||' de la ligne
				printf("\033[%im||\033[0m", (s % 2 ? 93 : 96)); // '||' du set
				
				if (compare_floats(arr0[pos], arr1[pos], COMPARE_DEEPH)) 
					printf("|| %i |  \033[42m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
				else
					printf("|| %i |  \033[41m %f --- %f \033[0m \n", i, arr0[pos], arr1[pos]);
			}
		}
	}
};

//=======================================================================================

void train_compare_weights(Train_t * train, float * with_this) {
	uint sets = train->sets;
	uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	//uint lines = train->data->lines;

	float * tmpt = (float*)malloc(sizeof(float) * (sets * weights));
	SAFE_CUDA(hipMemcpy(tmpt, train->_weight, sizeof(float) * (sets * weights), hipMemcpyDeviceToHost));
	wsep_compare_cpu_cpu(train->mdl->wsep, tmpt, with_this, sets, weights);
	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};

void train_compare_vars(Train_t * train, float * with_this) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * tmpt = (float*)malloc(sizeof(float) * (sets * lines * total));
	SAFE_CUDA(hipMemcpy(tmpt, train->_var, sizeof(float) * (sets * lines * total), hipMemcpyDeviceToHost));
	vsep_compare_cpu_cpu(train->mdl->vsep, tmpt, with_this, sets, lines, total);
	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};

void train_compare_locds(Train_t * train, float * with_this) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * tmpt = (float*)malloc(sizeof(float) * (sets * lines * locds));
	SAFE_CUDA(hipMemcpy(tmpt, train->_locd, sizeof(float) * (sets * lines * locds), hipMemcpyDeviceToHost));
	lsep_compare_cpu_cpu(train->mdl->lsep, tmpt, with_this, sets, lines, locds);
	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};

void train_compare_grads(Train_t * train, float * with_this) {
	uint sets = train->sets;
	//uint weights = train->mdl->weights;
	uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	uint lines = train->data->lines;

	float * tmpt = (float*)malloc(sizeof(float) * (sets * lines * total));
	SAFE_CUDA(hipMemcpy(tmpt, train->_grad, sizeof(float) * (sets * lines * total), hipMemcpyDeviceToHost));
	vsep_compare_cpu_cpu(train->mdl->vsep, tmpt, with_this, sets, lines, total);
	free(tmpt);

	printf("             C/Cuda  ||| Python\n");
};

void train_compare_meands(Train_t * train, float * with_this) {
	uint sets = train->sets;
	uint weights = train->mdl->weights;
	//uint total = train->mdl->total;
	//uint locds = train->mdl->locds;
	//uint lines = train->data->lines;

	float * tmpt = (float*)malloc(sizeof(float) * (sets * weights));
	SAFE_CUDA(hipMemcpy(tmpt, train->_meand, sizeof(float) * (sets * weights), hipMemcpyDeviceToHost));
	wsep_compare_cpu_cpu(train->mdl->wsep, tmpt, with_this, sets, weights);
	free(tmpt);

	printf("           C/Cuda  ||| Python\n");
};
